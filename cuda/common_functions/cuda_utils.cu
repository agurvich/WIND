#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>
#include "cuda_utils.h"

__device__ int get_system_bid(){
    return blockIdx.z*gridDim.y + blockIdx.y;
}

__global__ void cudaRoutineFlat(int Neqn_p_sys, float * d_arr){
    printf("float: %d - %.3f\n",threadIdx.x,d_arr[threadIdx.x]);
}

__global__ void cudaRoutineFlatInt(int Neqn_p_sys, int * d_arr){
    printf("int: %d - %d\n",threadIdx.x,d_arr[threadIdx.x]);
}
__global__ void cudaRoutine(int Neqn_p_sys, float ** d_arr,int index){
    printf("%d - %.2f hello\n",threadIdx.x,d_arr[index][threadIdx.x]);
}

__global__ void printfCUDA(float * pointer){
    printf("%f value of cuda pointer \n",*pointer);
}

__global__ void printFloatArrayCUDA(float * pointer, int Narr){
    // safety in case it's called with a bunch of threads lol
    if (threadIdx.x == 0 && blockIdx.x == 0){
        for (int i = 0; i<Narr; i++){
            printf("%.2e \t",pointer[i]);
        }
        printf("\n");
    }
}

__global__ void checkCublasINFO(
    int * INFO,
    int * bool_flag,
    int Nsystems){
    // replace the values of v1 with the error
    int bid = get_system_bid();

    if (bid < Nsystems){
        if (INFO[bid]){
            *bool_flag = 1;
        }
    }
}

void checkCublasErrorState(int * INFO,int * d_INFO_bool,int INFO_bool,int Nsystems, dim3 ode_gridDim){
    checkCublasINFO<<<ode_gridDim,1>>>(INFO, d_INFO_bool,Nsystems);
    hipMemcpy(&INFO_bool,&d_INFO_bool,sizeof(int),hipMemcpyDeviceToHost);
    printf("INFO: %d \n",INFO_bool);
    INFO_bool = 0;
    hipMemcpy(d_INFO_bool,&INFO_bool,sizeof(int),hipMemcpyHostToDevice);
}

const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

