#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>
#include "cuda_utils.h"

__device__ int get_system_bid(){
    return blockIdx.z*gridDim.y + blockIdx.y;
}

__global__ void cudaRoutineFlat(int Neqn_p_sys, float * d_arr){
    printf("float: %d - %.3f\n",threadIdx.x,d_arr[threadIdx.x]);
}

__global__ void cudaRoutineFlatInt(int Neqn_p_sys, int * d_arr){
    printf("int: %d - %d\n",threadIdx.x,d_arr[threadIdx.x]);
}
__global__ void cudaRoutine(int Neqn_p_sys, float ** d_arr,int index){
    printf("%d - %.2f hello\n",threadIdx.x,d_arr[index][threadIdx.x]);
}

__global__ void printfCUDA(float * pointer){
    printf("%f value of cuda pointer \n",*pointer);
}

__global__ void printFloatArrayCUDA(float * pointer, int Narr){
    // safety in case it's called with a bunch of threads lol
    if (threadIdx.x == 0 && blockIdx.x == 0){
        for (int i = 0; i<Narr; i++){
            printf("%.2e \t",pointer[i]);
        }
        printf("\n");
    }
}

__global__ void checkCublasINFO(
    int * INFO,
    int * bool_flag,
    int Nsystems){
    // replace the values of v1 with the error
    int bid = get_system_bid();

    if (bid < Nsystems){
        if (INFO[bid]){
            *bool_flag = 1;
        }
    }
}

void checkCublasErrorState(int * INFO,int * d_INFO_bool,int INFO_bool,int Nsystems, dim3 ode_gridDim){
    checkCublasINFO<<<ode_gridDim,1>>>(INFO, d_INFO_bool,Nsystems);
    hipMemcpy(&INFO_bool,&d_INFO_bool,sizeof(int),hipMemcpyDeviceToHost);
    printf("INFO: %d \n",INFO_bool);
    INFO_bool = 0;
    hipMemcpy(d_INFO_bool,&INFO_bool,sizeof(int),hipMemcpyHostToDevice);
}

const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            printf("HIPBLAS_STATUS_SUCCESS\n");
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            printf("HIPBLAS_STATUS_NOT_INITIALIZED\n");
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            printf("HIPBLAS_STATUS_ALLOC_FAILED\n");
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            printf("HIPBLAS_STATUS_INVALID_VALUE\n");
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            printf("HIPBLAS_STATUS_ARCH_MISMATCH\n");
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            printf("HIPBLAS_STATUS_MAPPING_ERROR\n");
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            printf("HIPBLAS_STATUS_EXECUTION_FAILED\n");
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            printf("HIPBLAS_STATUS_INTERNAL_ERROR\n");
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}


__device__ int getGJETID(int iterations){
    //return blockIdx.x*blockDim.x+threadIdx.x;
    return iterations*blockDim.x + threadIdx.x;
}

__device__ int getGJEBID(){
    // assumes that gridDim.y = Nsystems, and blockDim.x = Neqn_p_sys
    //return blockIdx.z*gridDim.y + blockIdx.y;
    return blockIdx.x;//*blockDim.x;
}

__device__ void scaleRow(
    float * row_array, 
    int Ndim,
    int place_index, 
    float * scale_factor_out,
    bool read_scale_factor){

    if (read_scale_factor){
        *scale_factor_out = row_array[place_index];
    }

    int tid;
    for (int iterations=0; iterations<(Ndim/blockDim.x); iterations++){
        tid = getGJETID(iterations);
        if (tid < Ndim){
            row_array[tid]/=(*scale_factor_out);
        }
    }
    __syncthreads();

}


__device__ void subtractRows(
    float * target_row_array,
    float * row_array, 
    int Ndim,
    int place_index, 
    float * scale_factor_out,
    bool read_scale_factor){

    // were we passed a scale_factor to scale by 
    //  or should we read it from the column we're 
    //  zeroing? 
    if (read_scale_factor){
        *scale_factor_out = target_row_array[place_index];
    }

    int tid;
    // loop over a row
    for (int iterations=0; iterations<(Ndim/blockDim.x); iterations++){
        tid = getGJETID(iterations);
        if (tid < Ndim){
            target_row_array[tid] -= (*scale_factor_out)*row_array[tid];
        }
    }

    // make sure everyone finishes before moving on 
    __syncthreads();
}

__device__ void cudaBreakpoint(){
;
}

__device__ void gjeUFactor(
    float * d_this_matrix_flat,
    float * d_inverse_matrix_flat,
    int Ndim){

    // allocate a place to store row scale factors
    //  so that they may be applied to the d_inverse_matrix
    float d_this_row_scale_factor[1];

    // put this matrix into upper triangular form
    for (int row_i=0; row_i<Ndim;row_i++){
        scaleRow(
            d_this_matrix_flat + row_i*Ndim, // this row
            Ndim, // how many elements in row
            row_i, // which column am i dividing by
            d_this_row_scale_factor,
            true); 

        // apply the same transformation to the inverse
        scaleRow(
            d_inverse_matrix_flat + row_i*Ndim, // this row
            Ndim,
            NULL,// use provided scale factor
            d_this_row_scale_factor,
            false);

        for (int next_row_i=row_i+1; next_row_i < Ndim; next_row_i++){
            subtractRows(
                d_this_matrix_flat + next_row_i*Ndim,
                d_this_matrix_flat + row_i*Ndim,
                Ndim,
                row_i, // which column am I zeroing out
                d_this_row_scale_factor,
                true);

            subtractRows(
                d_inverse_matrix_flat + next_row_i*Ndim,
                d_inverse_matrix_flat + row_i*Ndim,
                Ndim,
                NULL, // use provided scale factor
                d_this_row_scale_factor,
                false);
        }
    }
}

__device__ void gjeLFactor(
    float * d_this_matrix_flat,
    float * d_inverse_matrix_flat,
    int Ndim){

    //  so that they may be applied to the d_inverse_matrix
    float d_this_row_scale_factor[1];

    int bri;
    int bnri;
    // put it into lower triangular form, start from the bottom
    for (int row_i=0; row_i<Ndim;row_i++){
        bri = Ndim - 1 - row_i;
        for (int next_row_i=row_i+1; next_row_i < Ndim; next_row_i++){
            bnri = Ndim - 1 - next_row_i;
            subtractRows(
                d_this_matrix_flat + bnri*Ndim,
                d_this_matrix_flat + bri*Ndim,
                Ndim,
                bri, // which column am I zeroing out
                d_this_row_scale_factor,
                true);

            subtractRows(
                d_inverse_matrix_flat + bnri*Ndim,
                d_inverse_matrix_flat + bri*Ndim,
                Ndim,
                NULL, // use provided scale factor
                d_this_row_scale_factor,
                false);
        }
    }
}

__device__ void setIdentity(
    float * d_inverse_matrix_flat,
    int Ndim){

    int tid;
    // loop over whole matrix to set zeros
    for (int iterations=0; iterations<(Ndim*Ndim/blockDim.x); iterations++){
        tid = getGJETID(iterations);
        if (tid < Ndim*Ndim){
            d_inverse_matrix_flat[tid]=0;    
        }
    }

    // loop over the diagonal
    for (int iterations=0; iterations<(Ndim/blockDim.x); iterations++){
        tid = getGJETID(iterations);
        if (tid < Ndim){
            d_inverse_matrix_flat[tid*(Ndim+1)] = 1.0;
        }
    }
}

__device__ void gjeInvertMatrix(
    float * d_this_matrix_flat,
    float * d_inverse_matrix_flat,
    int Ndim){

    // allocate a temporary inverse matrix
    //extern __shared__ float d_inverse_matrix_flat[];

    // generate an identity matrix in the shared inverse matrix 
    setIdentity(d_inverse_matrix_flat,Ndim);

    gjeUFactor(d_this_matrix_flat,d_inverse_matrix_flat,Ndim);

    gjeLFactor(d_this_matrix_flat,d_inverse_matrix_flat,Ndim);
        
    /*
    // copy the output back
    int tid;
    for (int iterations=0; iterations<(Ndim*Ndim/blockDim.x); iterations++){
        tid = getGJETID(iterations);
        if (tid < Ndim*Ndim){
            d_this_matrix_flat[tid] = d_inverse_matrix_flat[tid];
        }
    }
    */
}

__global__ void gjeInvertMatrixBatched(
    float * d_matricess_flat,
    float * d_inverse_matricess_flat,
    int Ndim,
    int Nbatch){

    int bid = getGJEBID();
    gjeInvertMatrix(
        d_matricess_flat + bid*Ndim*Ndim,
        d_inverse_matricess_flat + bid*Ndim*Ndim,
        Ndim);
}
