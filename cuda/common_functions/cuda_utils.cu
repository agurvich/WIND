#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>
#include "cuda_utils.h"

__device__ int get_system_bid(){
    return blockIdx.z*gridDim.y + blockIdx.y;
}

__global__ void cudaRoutineFlat(int offset, float * d_arr){
    for (int thread_index = 0; thread_index < blockDim.x; thread_index++){
        if (threadIdx.x == thread_index){
            printf("%.6f\t",
                d_arr[offset+threadIdx.x]);
        }
        __syncthreads();
    }
    if (threadIdx.x == 0){
        printf("\n");
    }
}

__global__ void cudaRoutineFlatInt(int Neqn_p_sys, int * d_arr){
    printf("int: %d - %d\n",threadIdx.x,d_arr[threadIdx.x]);
}
__global__ void cudaRoutine(int Neqn_p_sys, float ** d_arr,int index){
    printf("%d - %.2f hello\n",threadIdx.x,d_arr[index][threadIdx.x]);
}

__global__ void printfCUDA(float * pointer){
    printf("%f value of cuda pointer \n",*pointer);
}

__global__ void printFloatArrayCUDA(float * pointer, int Narr){
    // safety in case it's called with a bunch of threads lol
    if (threadIdx.x == 0 && blockIdx.x == 0){
        for (int i = 0; i<Narr; i++){
            printf("%.2e \t",pointer[i]);
        }
        printf("\n");
    }
}

__global__ void checkCublasINFO(
    int * INFO,
    int * bool_flag,
    int Nsystems){
    // replace the values of v1 with the error
    int bid = get_system_bid();

    if (bid < Nsystems){
        if (INFO[bid]){
            *bool_flag = 1;
        }
    }
}

void checkCublasErrorState(int * INFO,int * d_INFO_bool,int INFO_bool,int Nsystems, dim3 ode_gridDim){
    checkCublasINFO<<<ode_gridDim,1>>>(INFO, d_INFO_bool,Nsystems);
    hipMemcpy(&INFO_bool,&d_INFO_bool,sizeof(int),hipMemcpyDeviceToHost);
    printf("INFO: %d \n",INFO_bool);
    INFO_bool = 0;
    hipMemcpy(d_INFO_bool,&INFO_bool,sizeof(int),hipMemcpyHostToDevice);
}

const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            printf("HIPBLAS_STATUS_SUCCESS\n");
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            printf("HIPBLAS_STATUS_NOT_INITIALIZED\n");
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            printf("HIPBLAS_STATUS_ALLOC_FAILED\n");
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            printf("HIPBLAS_STATUS_INVALID_VALUE\n");
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            printf("HIPBLAS_STATUS_ARCH_MISMATCH\n");
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            printf("HIPBLAS_STATUS_MAPPING_ERROR\n");
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            printf("HIPBLAS_STATUS_EXECUTION_FAILED\n");
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            printf("HIPBLAS_STATUS_INTERNAL_ERROR\n");
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

__device__ void cudaBreakpoint(){
;
}
