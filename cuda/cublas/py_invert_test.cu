// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010
 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
//#include <cusolverDn.h>


//void find_stepsizes(){}
//void calculate_derivative(){}
//void calculate_Jacobian(){}

//void SIE_step(
//  float * d_hs, //step-sizes to take
//  float ** d_Jacobianss, // a list of flattened jacobians nsystems x (ndim*ndim)
//  float ** d_derivativess, // a list of derivative vectors, nsystems x ndim
//  float ** d_yss, // a list of state vectors, nsystems x ndim
//  float ** d_identity, // the identity matrix, ideally stored in constant memory!!
//  int nsystems, // the number of ODE systems
//  int ndim, // the number of equations in each ODE
//  ){
//  /* Uses the semi-implicit backwards euler method to step multiple systems by h simultaneously:
//      y_n+1 = y_n + h(1 - hJ)^-1 f_n
//      
//  */ 
// -------------- invert the matrix -------------- */

//  // TODO pretty sure i need a multidimensional grid here, 
//  // blocks can't be 160x160 threads
//  // TODO have to have multiple hs here
//  //addArrayToBatchArrays<<<nsystems,ndim*ndim>>>(d_identity,d_Jacobianss,1.0,-h);
//  addArrayToBatchArraysVaryScale<<<nsystems,ndim*ndim>>>(d_identity,d_Jacobianss,-1.0,d_hs);
//  //cudaRoutine<<<1,ndim*ndim>>>(ndim,d_Jacobianss,0);

//  // host call to cublas, does LU factorization for matrices in d_Jacobianss, stores the result in... P? 
//  // the permutation array seems to be important for some reason
//  // but it is "batching" the call, it's good for inverting a bunch of small matrices where setup
//  // could be expensive. Potentially this will be a problem for us? 
//  cublasSgetrfBatched(
//      handle, // cublas handle
//      ndim, // leading dimension of matrix ??
//      d_Jacobianss, // matrix to factor
//      ndim, // number of rows&columns in matrix
//      P, // permutation matrix
//      INFO, // cublas status object
//      nsystems); // number of batches

//  // second cublas call, this one solves AX=B with B the inverse. It puts X in C_d
//  cublasSgetriBatched(
//      handle, // cublas handle
//      ndim, // leading dimension of matrix ??
//      (const float **)d_Jacobianss,
//      ndim,
//      P,
//      C_d,
//      ndim,
//      INFO,
//      nsystems);
// ----------------------------------------------- */

void printArray(int * arr,int N){
    for (int i = 0; i<N;i++){
        printf("%d ",arr[i]);
    }
    printf("\n");
}

void printFArray(float * arr, int N){
    for (int i = 0; i<N;i++){
        printf("%.2f ",arr[i]);
    }
    printf("\n");
}

__global__ void cudaRoutineFlat(int ndim, float * d_arr){
    printf("%d - %.2f hello\n",threadIdx.x,d_arr[threadIdx.x]);
}
__global__ void cudaRoutine(int ndim, float ** d_arr,int index){
    printf("%d - %.2f hello\n",threadIdx.x,d_arr[index][threadIdx.x]);
}

void setIdentityMatrix(float * identity,int ndim){
    for (int i=0; i<(ndim*ndim);i++){
        if (!(i%(ndim+1))){
            identity[i]=1;
        }
        else{
            identity[i]=0;
        }
    }
}

float ** initializeDeviceMatrix(float * h_flat, float ** p_d_flat, int arr_size,int nbatch){
    // returns a device pointer to the 2d array, the pointer to the 
    // flat array is the first element of the 2d array, just ask for 
    // more bytes
    
    // allocate device pointers

    float ** d_arr, *d_flat;
    hipMalloc(&d_arr,nbatch*sizeof(float *));
    hipMalloc(&d_flat, arr_size*nbatch*sizeof(float));

    // create a temporary array that partitions d_flat
    float **temp = (float **) malloc(nbatch*sizeof(float *));

    // arrange the array in column major order
    temp[0]=d_flat;
    for (int i=1; i<nbatch; i++){
        temp[i]=temp[i-1]+(arr_size);
    }

    // copy the temporary pointer's values to the device
    hipMemcpy(d_arr,temp,nbatch*sizeof(float *),hipMemcpyHostToDevice);

    // copy the actual values across
    hipMemcpy(d_flat,h_flat,arr_size*nbatch*sizeof(float),hipMemcpyHostToDevice);
    //cudaRoutine<<<1,9>>>(9,d_arr,0);

    // return what we want
    *p_d_flat=d_flat;
    return d_arr;
}

__global__ void addArrayToBatchArrays(float ** single_arr, float ** batch_arrs, float alpha, float beta){
    // assumes that gridDim = nbatch and blockDim = ndim
    batch_arrs[blockIdx.x][threadIdx.x]=alpha*single_arr[0][threadIdx.x]+ beta*batch_arrs[blockIdx.x][threadIdx.x];
}

/*float * getDFlatPointer(float ** d_arr,int nbatch){
    float * d_flat;
    float **temp=(float *)malloc(sizeof(float *))
    cudaMemcpy(d_arr,temp,nbatch*sizeof(float *),cudaMemcpyDeviceToHost);
    d_flat = temp[0];
    return d_flat;
    }
*/

void SIE_step(
    float ** d_Jacobianss, 
    float ** C_d,
    float ** d_identity,
    float ** d_derivatives,
    float * d_derivatives_flat,
    float * d_out_flat,
    int batchSize,
    int ndim){

/* -------------- initialize cublas -------------- */
    // initialize cublas status tracking pointers
    hipblasHandle_t handle;
    int *P, *INFO;

    // handle is something that connects cublas calls within a stream... something about v2 and 
    // being able to pass scalars by reference instead of by value. I don't really understand it
    // place to store cublas status stuff. 
    hipblasCreate(&handle);
    hipMalloc(&P, ndim * batchSize * sizeof(int));
    hipMalloc(&INFO,  batchSize * sizeof(int));
/* ----------------------------------------------- */

/* -------------- invert the matrix -------------- */

    // TODO pretty sure i need a multidimensional grid here, 
    // blocks can't be 160x160 threads
    float h = 1.0;
    addArrayToBatchArrays<<<batchSize,ndim*ndim>>>(d_identity,d_Jacobianss,1.0,-h);
    //cudaRoutine<<<1,ndim*ndim>>>(ndim,d_Jacobianss,0);

    // host call to cublas, does LU factorization for matrices in d_Jacobianss, stores the result in... P? 
    // the permutation array seems to be important for some reason
    // but it is "batching" the call, it's good for inverting a bunch of small matrices where setup
    // could be expensive. Potentially this will be a problem for us? 
    hipblasSgetrfBatched(handle,ndim,d_Jacobianss,ndim,P,INFO,batchSize);

    // second cublas call, this one solves AX=B with B the inverse. It puts X in C_d
    hipblasSgetriBatched(handle,ndim,(const float **)d_Jacobianss,ndim,P,C_d,ndim,INFO,batchSize);
/* ----------------------------------------------- */

/* -------------- perform a vector mult ---------- */
    
    // scalars for adding/multiplying
    float alpha = 1.0;
    float beta = 0.0;

    // define the identity matrix on the host
    float *many_identity_flat = (float *)malloc(batchSize*ndim*ndim*sizeof(float));
    for (int i=0; i<batchSize; i++){
        setIdentityMatrix(many_identity_flat+ndim*ndim*i,ndim);
    }
    
    // set a batchsize of one

    hipblasSgemmBatched(
        handle,// cublas handle
        HIPBLAS_OP_N,// no transformation
        HIPBLAS_OP_N,// no transformation
        ndim, //m- number of rows in A (and C)
        1, //n- number of columns in B (and C)
        ndim, //k-number of columns in A and rows in B
        (const float *) &alpha, // alpha scalar
        (const float **) C_d, // A matrix
        ndim, // leading dimension of the 2d array storing A??
        (const float **) d_derivatives, // B matrix (or n x 1 column vector)
        ndim, // leading dimension of the 2d array storing B??
        (const float *) &beta, // beta scalar
        (float **) d_derivatives, // output "matrix," let's overwrite B
        ndim, // leading dimension of the 2d array storing C??
        batchSize); // batch count
            
/* ----------------------------------------------- */

/* -------------- perform a vector addition ------ */
        // scale the dy vectors by the timestep size
        //scaleVector<<<batchSize,ndim>>>(d_derivatives_flat,hs);
        hipblasSaxpy(
            handle, // cublas handle
            ndim*batchSize, // number of elements in each vector
            (const float *) &alpha, // alpha scalar
            (const float *) d_derivatives_flat, // vector we are adding, flattened derivative vector
            1, // stride between consecutive elements
            d_out_flat, // vector we are replacing
            1); // stride between consecutive elements
/* ----------------------------------------------- */

    cudaRoutineFlat<<<1,ndim*batchSize>>>(ndim,d_out_flat);

    // shut down cublas
    hipFree(P); hipFree(INFO); hipblasDestroy(handle);
}

void invertMatrix(int batchSize,float * src_flat,int ndim){
    /*
    int blocksize,gridsize;
    if (ndim*ndim*batchSize < 1024){
        blocksize = ndim*ndim*batchSize;
        gridsize = 1;
    }
    else{
        blocksize = 1024;
        gridsize = ndim*ndim*batchSize/1024+1;
    }

    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( gridsize, 1 );
    */

    printf("Received %d arrays, each %d x %d:\n",batchSize,ndim,ndim);
    float **src = (float **)malloc(batchSize*sizeof(float *));
    src[0] = src_flat;
    for (int i=1; i<batchSize; i++){
        src[i] = src[i-1]+i*ndim*ndim;
    }

    //float *dest = (float *)malloc(batchSize*ndim*ndim*sizeof(float *));
    float *dest = src_flat;

    // define the identity matrix on the host
    float *identity_flat = (float *)malloc(ndim*ndim*sizeof(float));
    setIdentityMatrix(identity_flat,ndim);
    
    // set a batchsize of one
    float * d_identity_flat;
    float ** d_identity = initializeDeviceMatrix(identity_flat,&d_identity_flat,ndim*ndim,1);
    
/* -------------- move data to device ------------ */
    // allocate memory for matrices as a single "batch"
    float *d_Jacobianss_flat;
    float **d_Jacobianss = initializeDeviceMatrix(src_flat,&d_Jacobianss_flat,ndim*ndim,batchSize);

    float * my_vecs = (float *) malloc(batchSize*ndim*sizeof(float));

    for (int i=0; i<ndim*batchSize; i++){
        my_vecs[i]=i;
    }   

    // input derivative vectors
    float *d_derivatives_flat;
    float **d_derivatives = initializeDeviceMatrix(my_vecs,&d_derivatives_flat,ndim,batchSize);

    // output zero vectors
    float * zeros = (float *) malloc(batchSize*ndim*sizeof(float));
    for (int i=0; i<ndim*batchSize; i++){
        zeros[i]=0;
    }   

    float *d_out_flat;
    float **d_out = initializeDeviceMatrix(zeros,&d_out_flat,ndim,batchSize);

/* ----------------------------------------------- */

    
/* -------------- main integration loop ---------- */
    for (int i=0; i<1; i++){
        SIE_step(
            d_Jacobianss, // matrix (jacobian) input
            d_Jacobianss, // inverse output, overwrite d_Jacobianss
            d_identity, // pointer to identity (ideally in constant memory?)
            d_derivatives, // vector (derivatives) input
            d_derivatives_flat, // dy vector output
            d_out_flat, // y vector output
            batchSize, // number of systems
            ndim); // number of equations in each system
    }
/* ----------------------------------------------- */
    
/* -------------- copy data to host -------------- */
    // copy results to the destination array
    for (int i = 0; i < batchSize; i++){
      hipMemcpy((void *)(dest+i*ndim*ndim), d_Jacobianss_flat + (i*ndim*ndim), ndim*ndim*sizeof(float), hipMemcpyDeviceToHost);
    }
/* ----------------------------------------------- */

/* -------------- shutdown cublas   -------------- */
    hipFree(d_Jacobianss); //cudaFree(d_Jacobianssflat); free(A);
/* ----------------------------------------------- */

}
