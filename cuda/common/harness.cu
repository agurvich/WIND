#include <stdio.h>
#include "config.h"
#include "ode.h"


void * d_p_RHS_input; 

int cudaIntegrateSystem(
    float tnow, // the current time
    float tend, // the time we integrating the system to
    int n_integration_steps,
    WindFloat * constants, // the constants for each system
    WindFloat * equations, // a flattened array containing the y value for each equation in each system
    int Nsystems, // the number of systems
    int Nequations_per_system, // the number of equations in each system
    float ABSOLUTE, // the absolute tolerance
    float RELATIVE){ // the relative tolerance

#ifdef LOUD
#ifdef SIE
    printf("SIE Received %d systems, %d equations per system\n",Nsystems,Nequations_per_system);
#else
    printf("RK2 Received %d systems, %d equations per system\n",Nsystems,Nequations_per_system);
#endif
#endif

    // copy the arrays over to the device
    int Nequations = Nsystems*Nequations_per_system;
    long equations_size = Nequations*sizeof(WindFloat);

    WindFloat *constantsDevice;
    hipMalloc((void**)&constantsDevice, Nsystems*NUM_CONST*sizeof(WindFloat)); 
    hipMemcpy( constantsDevice, constants, Nsystems*NUM_CONST*sizeof(WindFloat), hipMemcpyHostToDevice ); 

    WindFloat *equationsDevice;
    hipMalloc((void**)&equationsDevice, equations_size); 
    hipMemcpy( equationsDevice, equations, equations_size, hipMemcpyHostToDevice ); 

#ifdef SIE
    WindFloat *JacobiansDevice;
    hipMalloc((void**)&JacobiansDevice, Nequations_per_system*equations_size); 
    //hipMemcpy( JacobiansDevice, Jacobians, equations_size, hipMemcpyHostToDevice ); 

    WindFloat *inversesDevice;
    hipMalloc((void**)&inversesDevice, Nequations_per_system*equations_size); 
    //hipMemcpy( JacobiansDevice, Jacobians, equations_size, hipMemcpyHostToDevice ); 
#endif

    int nloops=0;
    int * nloopsDevice;
    hipMalloc(&nloopsDevice, sizeof(int)); 
    hipMemcpy(nloopsDevice, &nloops, sizeof(int), hipMemcpyHostToDevice ); 


    float * tnowDevice;
    hipMalloc(&tnowDevice, sizeof(float)); 
    hipMemcpy(tnowDevice, &tnow, sizeof(float), hipMemcpyHostToDevice ); 

    float * tendDevice;
    hipMalloc(&tendDevice, sizeof(float)); 
    hipMemcpy(tendDevice, &tend, sizeof(float), hipMemcpyHostToDevice ); 

    // setup the grid dimensions

    //Maximum number of threads per multiprocessor:  2048
    //Maximum number of threads per block:           1024
    //Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
    //Max dimension size of a grid size    (x,y,z): (2.147.483.647, 65.535, 65.535)
    int blocksize,gridsize;
    if (Nequations_per_system < THREAD_BLOCK_LIMIT){
        blocksize = Nequations_per_system;
        gridsize = Nsystems;
    }
    else{
        printf("Too many equations/system, keep it below 1024\n");
        blocksize = 0;
        gridsize = 0;
    }

    //printf("%d blocksize, %d gridsize\n",blocksize,gridsize);
    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( gridsize, 1 );

    //shared mem -> 2 float arrays for each system and 1 shared flag
    integrateSystem<<<dimGrid,dimBlock,
        Nequations_per_system*(2*sizeof(WindFloat))+ sizeof(int)
        >>> (
        tnow, tend,
        (tend-tnow)/n_integration_steps,
        d_p_RHS_input,
        constantsDevice,equationsDevice,
#ifdef SIE
        JacobiansDevice,inversesDevice,
#else
        NULL,NULL,
#endif
        Nsystems,Nequations_per_system,
        nloopsDevice,
        ABSOLUTE,RELATIVE);
    
    // copy the new state back
    hipMemcpy(equations, equationsDevice, equations_size, hipMemcpyDeviceToHost ); 
    hipMemcpy(&nloops,nloopsDevice,sizeof(int),hipMemcpyDeviceToHost);
    //printf("c-equations after %.2f \n",equations[0]);

    // free up the memory on the device
    hipFree(constantsDevice);
    hipFree(equationsDevice);
#ifdef SIE
    hipFree(JacobiansDevice);hipFree(inversesDevice);
#endif
    hipFree(tendDevice);
    hipFree(tnowDevice);
    hipFree(nloopsDevice);

    // return how many steps were taken
    return nloops;
} // cudaIntegrateRK2

extern "C" {
    int WINDIntegrateSystem(
        float tnow, // the current time
        float tend, // the time we integrating the system to
        int n_integration_steps,
        WindFloat * constants, // the constants for each system
        WindFloat * equations, // a flattened array containing the y value for each equation in each system
        int Nsystems, // the number of systems
        int Nequations_per_system, // the number of equations in each system
        float ABSOLUTE, // the absolute tolerance
        float RELATIVE){

        return cudaIntegrateSystem(
            tnow,
            tend,
            n_integration_steps,
            constants,
            equations,
            Nsystems,
            Nequations_per_system,
            ABSOLUTE,
            RELATIVE);
    }
}
