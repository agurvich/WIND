#include <stdio.h>
#include "config.h"
#include "ode.h"


void * RHS_input; 

int cudaIntegrateSystem(
    float tnow, // the current time
    float tend, // the time we integrating the system to
    int n_integration_steps,
    WindFloat * constants, // the constants for each system
    WindFloat * equations, // a flattened array containing the y value for each equation in each system
    int Nsystems, // the number of systems
    int Nequations_per_system, // the number of equations in each system
    float ABSOLUTE, // the absolute tolerance
    float RELATIVE){ // the relative tolerance

#ifdef LOUD
#ifdef SIE
    printf("SIE Received %d systems, %d equations per system\n",Nsystems,Nequations_per_system);
#else
    printf("RK2 Received %d systems, %d equations per system\n",Nsystems,Nequations_per_system);
#endif
#endif

    // copy the arrays over to the device
    int Nequations = Nsystems*Nequations_per_system;
    long equations_size = Nequations*sizeof(WindFloat);

    WindFloat *constantsDevice;
    hipMalloc((void**)&constantsDevice, Nsystems*NUM_CONST*sizeof(WindFloat)); 
    hipMemcpy( constantsDevice, constants, Nsystems*NUM_CONST*sizeof(WindFloat), hipMemcpyHostToDevice ); 

    WindFloat *equationsDevice;
    hipMalloc((void**)&equationsDevice, equations_size); 
    hipMemcpy( equationsDevice, equations, equations_size, hipMemcpyHostToDevice ); 

#ifdef SIE
    WindFloat *JacobiansDevice;
    hipMalloc((void**)&JacobiansDevice, Nequations_per_system*equations_size); 
    //hipMemcpy( JacobiansDevice, Jacobians, equations_size, hipMemcpyHostToDevice ); 

    WindFloat *inversesDevice;
    hipMalloc((void**)&inversesDevice, Nequations_per_system*equations_size); 
    //hipMemcpy( JacobiansDevice, Jacobians, equations_size, hipMemcpyHostToDevice ); 
#endif

    int nloops=0;
    int * nloopsDevice;
    hipMalloc(&nloopsDevice, sizeof(int)); 
    hipMemcpy(nloopsDevice, &nloops, sizeof(int), hipMemcpyHostToDevice ); 


    float * tnowDevice;
    hipMalloc(&tnowDevice, sizeof(float)); 
    hipMemcpy(tnowDevice, &tnow, sizeof(float), hipMemcpyHostToDevice ); 

    float * tendDevice;
    hipMalloc(&tendDevice, sizeof(float)); 
    hipMemcpy(tendDevice, &tend, sizeof(float), hipMemcpyHostToDevice ); 

    // setup the grid dimensions
    int blocksize,gridsize;
    if (Nequations_per_system < THREAD_BLOCK_LIMIT){
        blocksize = Nequations_per_system;
        gridsize = Nsystems;
    }
    else{
        blocksize = THREAD_BLOCK_LIMIT;
        gridsize = Nequations/THREAD_BLOCK_LIMIT+1;
    }

    //printf("%d blocksize, %d gridsize\n",blocksize,gridsize);
    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( gridsize, 1 );


    read_texture<<<1,1>>>(RHS_input);
    hipDeviceSynchronize();

    //shared mem -> 2 float arrays for each system and 1 shared flag
    integrateSystem<<<dimGrid,dimBlock,
        Nequations_per_system*(2*sizeof(WindFloat))+ sizeof(int)
        >>> (
        tnow, tend,
        (tend-tnow)/n_integration_steps,
        constantsDevice,equationsDevice,
#ifdef SIE
        JacobiansDevice,inversesDevice,
#endif
        Nsystems,Nequations_per_system,
        nloopsDevice,
        ABSOLUTE,RELATIVE);
    
    // copy the new state back
    hipMemcpy(equations, equationsDevice, equations_size, hipMemcpyDeviceToHost ); 
    hipMemcpy(&nloops,nloopsDevice,sizeof(int),hipMemcpyDeviceToHost);
    //printf("c-equations after %.2f \n",equations[0]);

    // free up the memory on the device
    hipFree(constantsDevice);
    hipFree(equationsDevice);
#ifdef SIE
    hipFree(JacobiansDevice);hipFree(inversesDevice);
#endif
    hipFree(tendDevice);
    hipFree(tnowDevice);
    hipFree(nloopsDevice);

    // return how many steps were taken
    return nloops;
} // cudaIntegrateRK2

extern "C" {
    int WINDIntegrateSystem(
        float tnow, // the current time
        float tend, // the time we integrating the system to
        int n_integration_steps,
        WindFloat * constants, // the constants for each system
        WindFloat * equations, // a flattened array containing the y value for each equation in each system
        int Nsystems, // the number of systems
        int Nequations_per_system, // the number of equations in each system
        float ABSOLUTE, // the absolute tolerance
        float RELATIVE){

        return cudaIntegrateSystem(
            tnow,
            tend,
            n_integration_steps,
            constants,
            equations,
            Nsystems,
            Nequations_per_system,
            ABSOLUTE,
            RELATIVE);
    }
}
