#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include "explicit_solver.h"

#define ABSOLUTE_TOLERANCE 1e-6
#define RELATIVE_TOLERANCE 1e-6

__device__ float calculate_dydt(
    float tnow,
    float * constants,
    float * equations){
    // constraint equation, ne = nH+ + nHe+ + 2*nHe++
    float ne = equations[1]+equations[3]+equations[4]*2.0;

    /* constants = [
        Gamma_(e,H0), Gamma_(gamma,H0), 
        alpha_(H+),
        Gamma_(e,He0), Gamma_(gamma,He0), 
        Gamma_(e,He+), Gamma_(gamma,He+),
        alpha_(He+),
        alpha_(d),
        alpha_(He++)
        ] 
    */

    if (threadIdx.x == 0){
        // H0 : alpha_(H+) ne nH+ - (Gamma_(e,H0)ne + Gamma_(gamma,H0))*nH0
        return constants[2]*ne*equations[1]
            -(constants[0]*ne + constants[1])*equations[0]; 
    }
    else if (threadIdx.x == 1){
        // H+ : (Gamma_(e,H0)ne + Gamma_(gamma,H0))*nH0 - alpha_(H+) ne nH+
        return -constants[2]*ne*equations[1]
            +(constants[0]*ne + constants[1])*equations[0]; 
    }
    else if (threadIdx.x == 2){
        // He0 :(alpha_(He+)+alpha_(d)) ne nHe+ - (Gamma_(e,He0)ne + Gamma_(gamma,He0)) nHe0
        return (constants[7]+constants[8])*ne*equations[3] 
            - (constants[3]*ne+constants[4])*equations[2];
    }
    else if (threadIdx.x == 3){
        // He+ : 
        //  alpha_(He++) ne nHe++ 
        //  + (Gamma_(e,He0)ne + Gamma_(gamma,He0)) nHe0
        //  - (alpha_(He+)+alpha_(d)) ne nHe+ 
        //  - (Gamma_(e,He+)ne + Gamma_(gamma,He+)) nHe+
        return constants[9]*ne*equations[4] 
            + (constants[3]*ne+constants[4])*equations[2]  
            - (constants[7]+constants[8])*ne*equations[3] 
            - (constants[5]*ne+constants[6])*equations[3];
    }
    else if (threadIdx.x == 4){
        // He++ : -alpha_(He++) ne nHe++
        return -constants[9]*ne*equations[4];
    }
    else{
        return NULL;
    }
} // calculate_dydt

__device__ float rk2_innerstep(
    float tnow, // the current time
    float tstop, // the time we want to stop
    float h, // the timestep to take
    float * constants, // the constants for each system
    float * shared_temp_equations, // place to store temporary equations
    int Nsystems, // the number of systems
    int Nequations_per_system){ // the number of equations in each system

    float dydt = 0;

    while (tnow < tstop){
        // limit step size based on remaining time
        h = fmin(tstop - tnow, h);

        //calculate the derivative for this equation
        dydt = calculate_dydt(
            tnow,
            constants,
            shared_temp_equations);

        // update value of temporary equations
        shared_temp_equations[threadIdx.x] += h*dydt;
        tnow+=h;

    } // while(tnow < tstop)
    return shared_temp_equations[threadIdx.x];
}// rk2_innerstep

__global__ void integrate_rk2(
    float tnow, // the current time
    float tend, // the time we integrating the system to
    float * constants, // the constants for each system
    float * equations, // a flattened array containing the y value for each equation in each system
    int Nsystems, // the number of systems
    int Nequations_per_system,
    int * nsteps){ // the number of equations in each system

    // unique thread ID , based on local ID in block and block ID
    int tid = threadIdx.x + ( blockDim.x * blockIdx.x);

    extern __shared__ float total_shared[];
    // total_shared is a pointer to the beginning of this block's shared
    //  memory. If we want to use multiple shared memory arrays we must
    //  manually offset them within that block and allocate enough memory
    //  when initializing the kernel (<<dimGrid,dimBlock,sbytes>>)
    int * shared_error_flag = (int *) &total_shared[0];
    float * shared_equations = (float *) &total_shared[1];
    float * shared_temp_equations = (float *) &shared_equations[Nequations_per_system];

    float y1,y2;
    float h = (tend-tnow);

    // ensure thread within limit
    if (tid < Nsystems*Nequations_per_system ) {
        // copy the y values to shared memory
        shared_equations[threadIdx.x] = equations[tid];
        *shared_error_flag = 0;
        __syncthreads();

        //printf("%d thread %d block\n",threadIdx.x,blockIdx.x);
        while (tnow < tend){
            // make sure we don't overintegrate
            h = fmin(tend-tnow,h);

            // now reset the temporary equations
            shared_temp_equations[threadIdx.x] = shared_equations[threadIdx.x];
            __syncthreads();

            y1 = rk2_innerstep(
                tnow, tnow+h,
                h,
                constants,
                shared_temp_equations,
                Nsystems, Nequations_per_system );
            
            // now reset the temporary equations
            shared_temp_equations[threadIdx.x] = shared_equations[threadIdx.x];
            __syncthreads();

            y2 = rk2_innerstep(
                tnow, tnow+h,
                h/2,
                constants,
                shared_temp_equations,
                Nsystems, Nequations_per_system );

            // determine if any equation is above the absolute or relative tolerances
            if(fabs(y2 - y1) > ABSOLUTE_TOLERANCE || fabs((y2-y1)/(2*y2-y1+1e-12)) > RELATIVE_TOLERANCE){
                *shared_error_flag = 1;
                }
            __syncthreads();

            if (*shared_error_flag){
                // refine and start over
                h/=2;
                *shared_error_flag = 0;
            } // if shared_error_flag
            else{
                (*nsteps)++;
                // accept this step and update the shared array
                //  using local extrapolation (see NR e:17.2.3)
                shared_equations[threadIdx.x] = 2*y2-y1;
                tnow+=h;

                // let's get a little more optimistic
                h*=2;
            }// if shared_error_flag -> else

            __syncthreads();

        }// while tnow < tend

        // copy the y values back to global memory
        equations[tid]=shared_equations[threadIdx.x];
        if (threadIdx.x == 1 && blockIdx.x == 0){
            printf("nsteps taken: %d - tnow: %.2f\n",*nsteps,tnow);
        }
    } // if tid < nequations
} //integrate_rk2
