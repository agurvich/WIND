#include "hip/hip_runtime.h"
#include <stdio.h>
#include "explicit_solver.h"

#define ABSOLUTE_TOLERANCE 1e-3
#define RELATIVE_TOLERANCE 1e-3

__device__ float calculate_dydt(
    float tnow,
    float * constants,
    float * equations){
    // assumes that constant and equation are pointers that start 
    //  at the beginning of the block's values.
    int tid = blockIdx.x*blockDim.x+threadIdx.x;

    return constants[tid]*tnow*tnow;
} // calculate_dydyt

__device__ float euler_innerstep(
    float tnow, // the current time
    float tstop, // the time we want to stop
    float h, // the timestep to take
    float * constants, // the constants for each system
    float * shared_temp_equations, // place to store temporary equations
    int Nsystems, // the number of systems
    int Nequations_per_system){ // the number of equations in each system

    float dydt = 0;

    while (tnow < tstop){
        // limit step size based on remaining time
        h = fmin(tstop - tnow, h);

        //calculate the derivative for this equation
        dydt = calculate_dydt(
            tnow,
            constants,
            shared_temp_equations);

        // update value of temporary equations
        shared_temp_equations[threadIdx.x] += h*dydt;
        tnow+=h;

    } // while(tnow < tstop)
    return shared_temp_equations[threadIdx.x];
}// euler_innerstep

__global__ void integrate_euler(
    float tnow, // the current time
    float tend, // the time we integrating the system to
    float * constants, // the constants for each system
    float * equations, // a flattened array containing the y value for each equation in each system
    int Nsystems, // the number of systems
    int Nequations_per_system,
    int * nsteps){ // the number of equations in each system

    // unique thread ID , based on local ID in block and block ID
    int tid = threadIdx.x + ( blockDim.x * blockIdx.x);

    extern __shared__ float total_shared[];
    // total_shared is a pointer to the beginning of this block's shared
    //  memory. If we want to use multiple shared memory arrays we must
    //  manually offset them within that block and allocate enough memory
    //  when initializing the kernel (<<dimGrid,dimBlock,sbytes>>)
    int * shared_error_flag = (int *) &total_shared[0];
    float * shared_equations = (float *) &total_shared[1];
    float * shared_temp_equations = (float *) &shared_equations[Nequations_per_system];

    float y1,y2;
    float h = (tend-tnow);

    // ensure thread within limit
    if (tid < Nsystems*Nequations_per_system ) {
        // copy the y values to shared memory
        shared_equations[threadIdx.x] = equations[tid];
        *shared_error_flag = 0;
        __syncthreads();

        //printf("%d thread %d block\n",threadIdx.x,blockIdx.x);
        while (tnow < tend){
            // make sure we don't overintegrate
            h = fmin(tend-tnow,h);

            // now reset the temporary equations
            shared_temp_equations[threadIdx.x] = shared_equations[threadIdx.x];
            __syncthreads();

            y1 = euler_innerstep(
                tnow, tnow+h,
                h,
                constants,
                shared_temp_equations,
                Nsystems, Nequations_per_system );
            
            // now reset the temporary equations
            shared_temp_equations[threadIdx.x] = shared_equations[threadIdx.x];
            __syncthreads();

            y2 = euler_innerstep(
                tnow, tnow+h,
                h/2,
                constants,
                shared_temp_equations,
                Nsystems, Nequations_per_system );

            *shared_error_flag = y2 - y1 > ABSOLUTE_TOLERANCE || (y2-y1)/(2*y2-y1) > RELATIVE_TOLERANCE;
            __syncthreads();

            (*nsteps)++;
            if (*shared_error_flag){
                // refine and start over
                h/=2;
            } // if shared_error_flag
            else{
                // accept this step and update the shared array
                //  using local extrapolation (see NR e:17.2.3)
                shared_equations[threadIdx.x] = 2*y2-y1;
                tnow+=h;

                // let's get a little more optimistic
                h*=2;
            }// if shared_error_flag -> else

            __syncthreads();

        }// while tnow < tend

        // copy the y values back to global memory
        equations[tid]=shared_equations[threadIdx.x];
        if (threadIdx.x == 1 && blockIdx.x == 0){
            printf("nsteps taken: %d - tnow: %.2f\n",*nsteps,tnow);
        }
    } // if tid < nequations
} //integrate_euler
