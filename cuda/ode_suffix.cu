void resetSystem(
    float ** d_derivatives,
    float * d_derivatives_flat,
    float ** d_Jacobianss,
    float * d_Jacobianss_flat,
    float * d_constants,
    float * d_current_state_flat,
    float * jacobian_zeros,
    int Nsystems,
    int Neqn_p_sys,
    float tnow){

    dim3 ode_gridDim;
    configureGrid(
        Nsystems,Neqn_p_sys,
        NULL,
        NULL,
        &ode_gridDim,
        NULL);


    if (d_derivatives_flat !=NULL){
        // evaluate the derivative function at tnow
        calculateDerivatives<<<ode_gridDim,1>>>(
            d_derivatives_flat,
            d_constants,
            d_current_state_flat,
            Nsystems,
            Neqn_p_sys,
            tnow);
    }

    if (d_Jacobianss_flat != NULL){
        // reset the jacobian, which has been replaced by (I-hJ)^-1
        hipMemcpy(
            d_Jacobianss_flat,jacobian_zeros,
            Nsystems*Neqn_p_sys*Neqn_p_sys*sizeof(float),
            hipMemcpyHostToDevice);

        calculateJacobians<<<ode_gridDim,1>>>(
            d_Jacobianss,
            d_constants,
            d_current_state_flat,
            Nsystems,
            Neqn_p_sys,
            tnow);
    }
}
