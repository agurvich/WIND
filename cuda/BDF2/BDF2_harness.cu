#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>

#include "implicit_solver.h"
#include "ode.h"
#include "utils.h"
#include "cuda_utils.h"
#include "vector_kernels.h"

//#include <hipsolver.h>
//#include "magmablas.h"

int BDF2SolveSystem(
    float tnow,
    float tend,
    float timestep,
    float ** d_Jacobianss, // matrix (jacobian) input
    float * d_Jacobianss_flat,
    float * jacobian_zeros,
    float ** d_identity, // pointer to identity (ideally in constant memory?)
    float ** d_derivatives, // vector (derivatives) input
    float * d_derivatives_flat, // dy vector output
    float * d_current_state_flat, // y vector output
    float * d_previous_state_flat,
    float ** d_intermediate, // matrix memory for intermediate calculation
    float * d_intermediate_flat,// flattened memory for intermediate calculation
    float * d_constants,
    int Nsystems, // number of systems
    int Neqn_p_sys){

    
    int nsteps = 1; 
    hipError_t cuda_error_code;
/* -------------- configure the grid  ------------ */
    int threads_per_block = min(Neqn_p_sys,MAX_THREADS_PER_BLOCK);
    int x_blocks_per_grid = 1+Neqn_p_sys/MAX_THREADS_PER_BLOCK;
    int y_blocks_per_grid = min(Nsystems,MAX_BLOCKS_PER_GRID);
    int z_blocks_per_grid = 1+Nsystems/MAX_BLOCKS_PER_GRID;

    dim3 vector_gridDim(
        x_blocks_per_grid,
        y_blocks_per_grid,
        z_blocks_per_grid);

/* ----------------------------------------------- */
    // copies the values of y(n) -> y(n-1)
    //  now that we don't need the "previous" step
    overwriteVector<<<vector_gridDim,threads_per_block>>>(
        d_current_state_flat,
        d_previous_state_flat,
        Nsystems,Neqn_p_sys);

    // evaluate the derivative and jacobian at 
    //  the current state
    resetSystem(
        d_derivatives,
        d_derivatives_flat,
        d_Jacobianss,
        d_Jacobianss_flat,
        d_constants,
        d_current_state_flat,
        jacobian_zeros,
        Nsystems,
        Neqn_p_sys,
        tnow);

    SIE_step(
        timestep, // Nsystems length vector for timestep to use
        d_Jacobianss, // matrix (jacobian) input
        d_Jacobianss, // inverse output, overwrite d_Jacobianss
        d_identity, // pointer to identity (ideally in constant memory?)
        d_derivatives, // vector (derivatives) input
        d_derivatives_flat, // dy vector output
        d_current_state_flat, // y vector output
        Nsystems, // number of systems
        Neqn_p_sys); // number of equations in each system

    tnow+=timestep;

/* ----------------------------------------------- */

    hipblasHandle_t handle;
    hipblasStatus_t error;
    hipblasCreate(&handle);
/* -------------- main integration loop ---------- */
    while (tnow < tend){
        nsteps++;
        
        // evaluate the derivative and jacobian at 
        //  the current state
        resetSystem(
            d_derivatives,
            d_derivatives_flat,
            d_Jacobianss,
            d_Jacobianss_flat,
            d_constants,
            d_current_state_flat,
            jacobian_zeros,
            Nsystems,
            Neqn_p_sys,
            tnow);

    /* -------------- perform the state switcheroo --- */
        
        //  (y(n)-y(n-1)) into d_intermediate_flat
        addVectors<<<vector_gridDim,threads_per_block>>>(
            -1.0,d_previous_state_flat,
            1.0, d_current_state_flat,
            d_intermediate_flat,Nsystems,Neqn_p_sys);

        // copies the values of y(n) -> y(n-1)
        //  now that we don't need the "previous" step
        overwriteVector<<<vector_gridDim,threads_per_block>>>(
            d_current_state_flat,
            d_previous_state_flat,Nsystems,Neqn_p_sys);
    /* ----------------------------------------------- */


        SIE_step(
            2.0/3.0*timestep, // Nsystems length vector for timestep to use
            d_Jacobianss, // matrix (jacobian) input
            d_Jacobianss, // inverse output, overwrite d_Jacobianss
            d_identity, // pointer to identity (ideally in constant memory?)
            d_derivatives, // vector (derivatives) input
            d_derivatives_flat, // dy vector output
            d_current_state_flat, // y vector output
            Nsystems, // number of systems
            Neqn_p_sys); // number of equations in each system


    /* -------------- perform two matrix-vector mults  */
        // multiply (I-2/3h*Js)^-1 x (y(n)-y(n-1)), 
        //  overwrite the output into d_intermediate

        float alpha = 1.0;
        float beta = 0.0;

        error = hipblasSgemmBatched(
            handle,// cublas handle
            HIPBLAS_OP_N,// no transformation
            HIPBLAS_OP_N,// no transformation
            Neqn_p_sys, //m- number of rows in A (and C)
            1, //n- number of columns in B (and C)
            Neqn_p_sys, //k-number of columns in A and rows in B
            (const float *) &alpha, // alpha scalar
            (const float **) d_Jacobianss, // has been replaced by 1-2/3h by most recent SIE_step
            Neqn_p_sys, // leading dimension of the 2d array storing A??
            (const float **) d_intermediate, // B matrix (or n x 1 column vector)
            Neqn_p_sys, // leading dimension of the 2d array storing B??
            (const float *) &beta, // beta scalar
            (float **) d_intermediate, // output "matrix," let's overwrite B
            Neqn_p_sys, // leading dimension of the 2d array storing C??
            Nsystems); // batch count

        addVectors<<<vector_gridDim,threads_per_block>>>(
            1.0/3.0,d_intermediate_flat,
            1.0, d_current_state_flat,
            d_current_state_flat,Nsystems,Neqn_p_sys);

        tnow+=timestep;

    }
    hipblasDestroy(handle);
    return nsteps;
}

int BDF2ErrorLoop(
    float tnow,
    float tend,
    float ** d_Jacobianss, // matrix (jacobian) input
    float * d_Jacobianss_flat,
    float * jacobian_zeros,
    float ** d_identity, // pointer to identity (ideally in constant memory?)
    float ** d_derivatives, // vector (derivatives) input
    float * d_derivatives_flat, // dy vector output
    float * equations,
    float * d_current_state_flat, // y vector output
    float * d_half_current_state_flat,
    float * d_previous_state_flat,
    float ** d_intermediate,
    float * d_intermediate_flat,
    float * d_constants,
    int Nsystems, // number of systems
    int Neqn_p_sys){

    float timestep = tend-tnow;
    int n_integration_steps = 2;

    int * error_flag = (int *) malloc(sizeof(int));
    int * d_error_flag;
    hipMalloc(&d_error_flag,sizeof(int));
    *error_flag = 0;
    hipMemcpy(d_error_flag,error_flag,sizeof(int),hipMemcpyHostToDevice);

/* -------------- configure the grid  ------------ */
    int threads_per_block = min(Neqn_p_sys,MAX_THREADS_PER_BLOCK);
    int x_blocks_per_grid = 1+Neqn_p_sys/MAX_THREADS_PER_BLOCK;
    int y_blocks_per_grid = min(Nsystems,MAX_BLOCKS_PER_GRID);
    int z_blocks_per_grid = 1+Nsystems/MAX_BLOCKS_PER_GRID;

    dim3 vector_gridDim(
        x_blocks_per_grid,
        y_blocks_per_grid,
        z_blocks_per_grid);

/* ----------------------------------------------- */
    
    // use a flag as a counter, why not
    int unsolved = 1;
    int nsteps=0;
    while (unsolved){
        nsteps+= BDF2SolveSystem(
            tnow,
            tend,
            timestep/n_integration_steps,
            d_Jacobianss,
            d_Jacobianss_flat,
            jacobian_zeros,
            d_identity,
            d_derivatives,
            d_derivatives_flat,
            d_current_state_flat,
            d_previous_state_flat,
            d_intermediate, // matrix memory for intermediate calculation
            d_intermediate_flat,// flattened memory for intermediate calculation
            d_constants,
            Nsystems,
            Neqn_p_sys);

#ifdef ADAPTIVETIMESTEP 
        n_integration_steps*=2;

        nsteps+= BDF2SolveSystem(
            tnow,
            tend,
            timestep/n_integration_steps,
            d_Jacobianss,
            d_Jacobianss_flat,
            jacobian_zeros,
            d_identity,
            d_derivatives,
            d_derivatives_flat,
            d_half_current_state_flat,// the output state vector
            d_previous_state_flat,
            d_intermediate, // matrix memory for intermediate calculation
            d_intermediate_flat,// flattened memory for intermediate calculation
            d_constants,
            Nsystems,
            Neqn_p_sys);

        // determine if ANY of the INDEPENDENT systems are above the 
        //  the tolerance and fail them all. NOTE: this makes them not
        //  independent... 
        checkError<<<vector_gridDim,threads_per_block>>>(
            d_current_state_flat,d_half_current_state_flat,d_error_flag,
            Nsystems,Neqn_p_sys);

        // copy back the bool flag and determine if we done did it
        hipMemcpy(error_flag,d_error_flag,sizeof(int),hipMemcpyDeviceToHost);
        //*error_flag = 0;
        
        if (*error_flag){
            // increase the refinement level
            unsolved++;
            // put an upper limit on the refinement
            if (unsolved > 9){
                break;
            }

#ifdef LOUD
            printf("refining...%d\n",unsolved);
#endif
            *error_flag = 0;

            // reset the error flag on the device
            hipMemcpy(d_error_flag,error_flag,sizeof(int),hipMemcpyHostToDevice);
        
            // reset the equations
            hipMemcpy(
                d_current_state_flat,
                equations,
                Nsystems*Neqn_p_sys*sizeof(float),
                hipMemcpyHostToDevice);
            hipMemcpy(
                d_half_current_state_flat,
                equations,
                Nsystems*Neqn_p_sys*sizeof(float),
                hipMemcpyHostToDevice);
        }// if unsolved
        else{
            // we did it, let's exit the loop gracefully
            unsolved=0;
        }
#else
        // take only this one step and call it a day, simplest way to 
        //  quit early is to copy the values from d_equations_flat to d_half_equations_flat and
        //  return normally. 
        hipMemcpy(d_half_current_state_flat,d_current_state_flat,Nsystems*Neqn_p_sys*sizeof(float),hipMemcpyDeviceToDevice);
        unsolved=0;

#endif
    }// while unsolved

    // free up memory
    hipFree(d_error_flag);
    free(error_flag);

    // return computations performed
    return nsteps;
}

int cudaIntegrateBDF2(
    float tnow, // the current time
    float tend, // the time we integrating the system to
    float * constants, // the constants for each system
    float * equations, // a flattened array containing the y value for each equation in each system
    int Nsystems, // the number of systems
    int Neqn_p_sys){ // the number of equations in each system

#ifdef LOUD
    printf("BDF2 Received %d systems, %d equations per system\n",Nsystems,Neqn_p_sys);
#endif
    float *dest = equations;

    // define the identity matrix on the host
    float *identity_flat = (float *)malloc(Neqn_p_sys*Neqn_p_sys*sizeof(float));
    setIdentityMatrix(identity_flat,Neqn_p_sys);
    
    // set a batchsize of one
    float * d_identity_flat;
    float ** d_identity = initializeDeviceMatrix(identity_flat,&d_identity_flat,Neqn_p_sys*Neqn_p_sys,1);
    
/* -------------- move data to device ------------ */
    // zeros to initialize jacobians with
    float * jacobian_zeros = (float *) malloc(Nsystems*Neqn_p_sys*Neqn_p_sys*sizeof(float));
    for (int i=0; i<Neqn_p_sys*Neqn_p_sys*Nsystems; i++){
        jacobian_zeros[i]=0;
    }   

    // allocate memory for Jacobian matrices as a single "batch"
    float *d_Jacobianss_flat;
    float **d_Jacobianss = initializeDeviceMatrix(jacobian_zeros,&d_Jacobianss_flat,Neqn_p_sys*Neqn_p_sys,Nsystems);

    // initialize state-equation vectors
    float * zeros = (float *) malloc(Nsystems*Neqn_p_sys*sizeof(float));
    for (int i=0; i<Neqn_p_sys*Nsystems; i++){
        zeros[i]=0;
    }   

    // constants that define the ODEs
    /* TODO put this in constant memory instead-- does the below work? 
    __constant__ float d_constants[NUM_CONST]; // NUM_CONST #define'd in ode.h
    hipMemcpyToSymbol(HIP_SYMBOL(constants),d_constants,sizeof(d_constants));
    */
    float * d_constants;
    hipMalloc(&d_constants,NUM_CONST*sizeof(float));
    hipMemcpy(d_constants,constants,NUM_CONST*sizeof(float),hipMemcpyHostToDevice);

    // state equations, where output will be stored
    float *d_current_state_flat;
    float **d_current_state = initializeDeviceMatrix(equations,&d_current_state_flat,Neqn_p_sys,Nsystems);

    float *d_half_current_state_flat;
    float **d_half_current_state = initializeDeviceMatrix(
        equations,&d_half_current_state_flat,Neqn_p_sys,Nsystems);

    // saving previous step Y(n-1) because we need that for BDF2
    float *d_previous_state_flat;
    float **d_previous_state = initializeDeviceMatrix(zeros,&d_previous_state_flat,Neqn_p_sys,Nsystems);

    // memory for intermediate calculation... reuse it so we aren't constantly allocating
    //  and deallocating memory, NOTE can we remove this??
    float *d_intermediate_flat;
    float **d_intermediate = initializeDeviceMatrix(zeros,&d_intermediate_flat,Neqn_p_sys,Nsystems);

    // initialize derivative vectors
    float *d_derivatives_flat;
    float **d_derivatives = initializeDeviceMatrix(zeros,&d_derivatives_flat,Neqn_p_sys,Nsystems);

/* ----------------------------------------------- */

    int nsteps = BDF2ErrorLoop(
        tnow,
        tend,
        d_Jacobianss, // matrix (jacobian) input
        d_Jacobianss_flat,
        jacobian_zeros,
        d_identity, // pointer to identity (ideally in constant memory?)
        d_derivatives, // vector (derivatives) input
        d_derivatives_flat, // dy vector output
        equations,
        d_current_state_flat, // y vector output
        d_half_current_state_flat,
        d_previous_state_flat,
        d_intermediate, // matrix memory for intermediate calculation
        d_intermediate_flat,// flattened memory for intermediate calculation
        d_constants,
        Nsystems, // number of systems
        Neqn_p_sys);
    
#ifdef LOUD
    printf("nsteps taken: %d - tnow: %.2f\n",nsteps,tend);
#endif



/* -------------- copy data to host -------------- */
    // retrieve the output
    hipMemcpy(dest, d_half_current_state_flat, Neqn_p_sys*Nsystems*sizeof(float), hipMemcpyDeviceToHost);
/* ----------------------------------------------- */

/* -------------- shutdown by freeing memory   --- */
    hipFree(d_identity); hipFree(d_identity_flat);
    hipFree(d_Jacobianss); hipFree(d_Jacobianss_flat);
    hipFree(d_current_state); hipFree(d_current_state_flat);
    hipFree(d_half_current_state); hipFree(d_half_current_state_flat);
    hipFree(d_previous_state); hipFree(d_previous_state_flat);
    hipFree(d_intermediate); hipFree(d_intermediate_flat);
    hipFree(d_derivatives); hipFree(d_derivatives_flat);

    free(zeros); free(jacobian_zeros);
    free(identity_flat);
/* ----------------------------------------------- */
    //return how many steps were taken
    return nsteps;
}
