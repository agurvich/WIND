#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>

#include "implicit_solver.h"
#include "ode.h"
#include "utils.h"
#include "cuda_utils.h"
#include "vector_kernels.h"

//#include <hipsolver.h>
//#include "magmablas.h"

//#define COMMENTBDF2

void BDF2_step(
    float timestep, // device pointer to the current timestep (across all systems, lame!!)
    float ** d_Jacobianss,  // Nsystems x Neqn_p_sys*Neqn_p_sys 2d array with flattened jacobians
    float ** d_inverse, // Nsystems x Neqn_p_sys*Neqn_p_sys 2d array to store output (same as jacobians to overwrite)
    float ** d_identity, // 1 x Neqn_p_sys*Neqn_p_sys array storing the identity (ideally in constant memory?)
    float ** d_derivatives, // Nsystems x Neqn_p_sys 2d array to store derivatives
    float * d_derivatives_flat, // Nsystems*Neqn_p_sys 1d array (flattened above)
    float * d_previous_state_flat, // state vector from previous timestep
    float * d_current_state_flat, // state vector from this timestep, where output goes
    float ** d_intermediate, // matrix holding intermediate values used internally for the calculation
    float * d_intermediate_flat, // flat array storing intermediate values used internally for the calculation
    int Nsystems, // number of ODE systems
    int Neqn_p_sys){ // number of equations in each system

#ifndef COMMENTBDF2
/* -------------- initialize cublas -------------- */
    // initialize cublas status tracking pointers
    hipblasHandle_t handle;
    int *P, *INFO;
    // handle is something that connects cublas calls within a stream... something about v2 and 
    // being able to pass scalars by reference instead of by value. I don't really understand it
    // place to store cublas status stuff. 
    hipblasCreate(&handle);
    hipMalloc(&P, Neqn_p_sys * Nsystems * sizeof(int));
    hipMalloc(&INFO,  Nsystems * sizeof(int));

    //hipblasSetPointerMode(handle,HIPBLAS_POINTER_MODE_DEVICE);
/* ----------------------------------------------- */


/* -------------- calculate the timestep --------- */
    /*
    int * d_max_index;
    hipMalloc(&d_max_index,sizeof(int));
    float * d_alpha, * d_beta;
    hipMalloc(&d_alpha,sizeof(float));
    hipMalloc(&d_beta,sizeof(float));
    */

    // scalars for adding/multiplying
    float alpha = 1.0;
    //hipMemcpy(d_alpha,&alpha,sizeof(float),hipMemcpyHostToDevice);
    float beta = 2.0/3.0;
    //hipMemcpy(d_beta,&beta,sizeof(float),hipMemcpyHostToDevice);

    /*
    // TODO don't really understand how this should be working :|
    hipblasIsamax(
        handle, // cublas handle
        Nsystems*Neqn_p_sys, // number of elements in the vector
        d_derivatives_flat, // the vector to take the max of
        1, // the stride between elements of the vector
        d_max_index); // the index of the max element of the vector
    */

    // overwrite beta = 2/3 with beta = 0
    beta = 0.0;
    //hipMemcpy(d_beta,&beta,sizeof(float),hipMemcpyHostToDevice);
/* ----------------------------------------------- */

/* -------------- configure the grid  ------------ */
    int threads_per_block = min(Neqn_p_sys,MAX_THREADS_PER_BLOCK);
    int x_blocks_per_grid = 1+Neqn_p_sys/MAX_THREADS_PER_BLOCK;
    int y_blocks_per_grid = min(Nsystems,MAX_BLOCKS_PER_GRID);
    int z_blocks_per_grid = 1+Nsystems/MAX_BLOCKS_PER_GRID;

    dim3 matrix_gridDim(
        x_blocks_per_grid*Neqn_p_sys,
        y_blocks_per_grid,
        z_blocks_per_grid);

    dim3 vector_gridDim(
        x_blocks_per_grid,
        y_blocks_per_grid,
        z_blocks_per_grid);
/* ----------------------------------------------- */



/* -------------- invert the matrix -------------- */
    // compute (I-2/3hJ) with a custom kernel, here d_timestep = 2/3h
    addArrayToBatchArrays<<<matrix_gridDim,threads_per_block>>>(
        d_identity,
        d_Jacobianss,
        1.0,
        -1.0,
        timestep,
        Nsystems,Neqn_p_sys);

    // host call to cublas, does LU factorization for matrices in d_Jacobianss, stores the result in... P?
    // the permutation array seems to be important for some reason
    hipblasSgetrfBatched(
        handle, // cublas handle
        Neqn_p_sys, // leading dimension of A??
        d_Jacobianss, // matrix to factor, here I-hs*Js
        Neqn_p_sys, // 
        P, // permutation matrix
        INFO, // cublas status object
        Nsystems); // number of systems

    // second cublas call, this one solves AX=B with B the identity. It puts X in d_inverse
    hipblasSgetriBatched(
        handle, // cublas handle
        Neqn_p_sys, // leading dimension of A??
        (const float **)d_Jacobianss, // matrix to inverse, here I-hs*Js
        Neqn_p_sys, // leading dimension of B??
        P, // permutation matrix
        d_inverse, // output matrix
        Neqn_p_sys, // 
        INFO, // cublas status object
        Nsystems); // number of systems
/* ----------------------------------------------- */

/* -------------- perform the state switcheroo --- */
    //  (y(n)-y(n-1)) into d_intermediate_flat
    addVectors<<<vector_gridDim,threads_per_block>>>(
        -1.0,d_previous_state_flat,
        1.0, d_current_state_flat,
        d_intermediate_flat,Nsystems,Neqn_p_sys);

    // copies the values of y(n) -> y(n-1)
    //  now that we don't need the "previous" step
    overwriteVector<<<vector_gridDim,threads_per_block>>>(
        d_current_state_flat,
        d_previous_state_flat,Nsystems,Neqn_p_sys);
/* ----------------------------------------------- */

/* -------------- perform two matrix-vector mults  */
    // multiply (I-2/3h*Js)^-1 x (y(n)-y(n-1)), 
    //  overwrite the output into d_intermediate
    hipblasSgemmBatched(
        handle,// cublas handle
        HIPBLAS_OP_N,// no transformation
        HIPBLAS_OP_N,// no transformation
        Neqn_p_sys, //m- number of rows in A (and C)
        1, //n- number of columns in B (and C)
        Neqn_p_sys, //k-number of columns in A and rows in B
        (const float *) &alpha, // alpha scalar
        (const float **) d_inverse, // A matrix
        Neqn_p_sys, // leading dimension of the 2d array storing A??
        (const float **) d_intermediate, // B matrix (or n x 1 column vector)
        Neqn_p_sys, // leading dimension of the 2d array storing B??
        (const float *) &beta, // beta scalar
        (float **) d_intermediate, // output "matrix," let's overwrite B
        Neqn_p_sys, // leading dimension of the 2d array storing C??
        Nsystems); // batch count

    // multiply 2/3h*(I-2/3h*Js)^-1 x fs
    //  store the output in d_derivatives
    hipblasSgemmBatched(
        handle,// cublas handle
        HIPBLAS_OP_N,// no transformation
        HIPBLAS_OP_N,// no transformation
        Neqn_p_sys, //m- number of rows in A (and C)
        1, //n- number of columns in B (and C)
        Neqn_p_sys, //k-number of columns in A and rows in B
        (const float *) &timestep, // alpha scalar
        (const float **) d_inverse, // A matrix
        Neqn_p_sys, // leading dimension of the 2d array storing A??
        (const float **) d_derivatives, // B matrix (or n x 1 column vector)
        Neqn_p_sys, // leading dimension of the 2d array storing B??
        (const float *) &beta, // beta scalar
        (float **) d_derivatives, // output "matrix," let's overwrite B
        Neqn_p_sys, // leading dimension of the 2d array storing C??
        Nsystems); // batch count

    // add 1/3(I-2/3hJ)^-1(Y(n)-Y(n-1)) 
    //  to Y(n) (from d_previous_state_flat), 
    //  storing the output in d_current_state_flat
    addVectors<<<vector_gridDim,threads_per_block>>>(
        1.0, d_previous_state_flat,
        1.0/3.0, d_intermediate_flat,
        d_current_state_flat,Nsystems,Neqn_p_sys);

    // add [Y(n) + 1/3(I-2/3hJ)^-1(Y(n)-Y(n-1))] 
    //  to [2/3h*(I-2/3hJ)^-1 x f] to get BDF 2 sln,
    //  storing the output in  d_current_state_flat
    addVectors<<<vector_gridDim,threads_per_block>>>(
        1.0, d_current_state_flat, 
        1.0, d_derivatives_flat,  // only need 1.0 here because d_timestep is representing 2/3h
        d_current_state_flat,Nsystems,Neqn_p_sys);

/* ----------------------------------------------- */

/* -------------- perform a vector addition ------ */
    // scale the dy vectors by the timestep size
    //scaleVector<<<vector_gridDim,threads_per_block>>>(d_derivatives_flat,d_timesteps,Nystems,Neqn_p_sys);
    
    /*
    // add ys + h x dys = ys + h x [(I-h*Js)^-1*fs]
    hipblasSaxpy(
        handle, // cublas handle
        Neqn_p_sys*Nsystems, // number of elements in each vector
        (const float *) d_timestep, // alpha scalar <-- can't use device pointer???
        (const float *) d_derivatives_flat, // vector we are adding, flattened derivative vector
        1, // stride between consecutive elements
        d_out_flat, // vector we are replacing
        1); // stride between consecutive elements
    //cudaRoutineFlat<<<1,Nsystems*Neqn_p_sys>>>(Neqn_p_sys,d_out_flat);
    */
/* ----------------------------------------------- */
    
    hipFree(P); hipFree(INFO); hipblasDestroy(handle);
    //hipFree(d_max_index); hipFree(d_alpha);hipFree(d_beta);

#endif
    // increment the timestep by whatever we just stepped by
    // allowing the device to vary/choose what it is (so we have
    // to copy it over). In FIXEDTIMESTEP mode this is silly but 
    // even still necessary.
    //float timestep = 1.0;
    //hipMemcpy(&timestep,d_timestep,sizeof(float),hipMemcpyDeviceToHost);
    
    // this changes the meaning of timestep to be 2/3 h instead of h 
    //  without actually changing any code.
    //*p_time+=3.0/2.0*timestep;

    // shut down cublas
    //TODO should free more stuff here?
}

int BDF2SolveSystem(
    float tnow,
    float tend,
    float timestep,
    float ** d_Jacobianss, // matrix (jacobian) input
    float * d_Jacobianss_flat,
    float * jacobian_zeros,
    float ** d_identity, // pointer to identity (ideally in constant memory?)
    float ** d_derivatives, // vector (derivatives) input
    float * d_derivatives_flat, // dy vector output
    float * d_current_state_flat, // y vector output
    float * d_previous_state_flat,
    float ** d_intermediate, // matrix memory for intermediate calculation
    float * d_intermediate_flat,// flattened memory for intermediate calculation
    float * d_constants,
    int Nsystems, // number of systems
    int Neqn_p_sys){

    
    int nsteps = 1;
    SIE_step(
        timestep, // Nsystems length vector for timestep to use
        d_Jacobianss, // matrix (jacobian) input
        d_Jacobianss, // inverse output, overwrite d_Jacobianss
        d_identity, // pointer to identity (ideally in constant memory?)
        d_derivatives, // vector (derivatives) input
        d_derivatives_flat, // dy vector output
        d_current_state_flat, // y vector output
        Nsystems, // number of systems
        Neqn_p_sys); // number of equations in each system

/* -------------- configure the grid  ------------ */
    int threads_per_block = min(Neqn_p_sys,MAX_THREADS_PER_BLOCK);
    int x_blocks_per_grid = 1+Neqn_p_sys/MAX_THREADS_PER_BLOCK;
    int y_blocks_per_grid = min(Nsystems,MAX_BLOCKS_PER_GRID);
    int z_blocks_per_grid = 1+Nsystems/MAX_BLOCKS_PER_GRID;

    dim3 vector_gridDim(
        x_blocks_per_grid,
        y_blocks_per_grid,
        z_blocks_per_grid);

    dim3 ode_gridDim(
        1,
        y_blocks_per_grid,
        z_blocks_per_grid);

/* ----------------------------------------------- */

    // copies the values of y(n) -> y(n-1)
    //  now that we don't need the "previous" step
    overwriteVector<<<vector_gridDim,threads_per_block>>>(
        d_current_state_flat,
        d_previous_state_flat,
        Nsystems,Neqn_p_sys);
    tnow+=timestep;
/* ----------------------------------------------- */

/* -------------- main integration loop ---------- */
    while (tnow < tend){
        nsteps++;
        
        // evaluate the derivative function at tnow
        calculateDerivatives<<<ode_gridDim,1>>>(
            d_derivatives_flat,
            d_constants,
            d_current_state_flat,
            Nsystems,
            Neqn_p_sys,
            tnow);

        // reset the jacobian, which has been replaced by (I-hJ)^-1
        hipMemcpy(
            d_Jacobianss_flat,jacobian_zeros,
            Nsystems*Neqn_p_sys*Neqn_p_sys*sizeof(float),
            hipMemcpyHostToDevice);

        calculateJacobians<<<ode_gridDim,1>>>(
            d_Jacobianss,
            d_constants,
            d_current_state_flat,
            Nsystems,
            Neqn_p_sys,
            tnow);

        BDF2_step(
            2.0/3.0*timestep, // Nsystems length vector for timestep to use
            d_Jacobianss, // matrix (jacobian) input
            d_Jacobianss, // inverse output, overwrite d_Jacobianss
            d_identity, // pointer to identity (ideally in constant memory?)
            d_derivatives, // vector (derivatives) input
            d_derivatives_flat, // dy vector output
            d_previous_state_flat,// Y(n-1) vector
            d_current_state_flat, // Y(n) vector output
            d_intermediate, // matrix memory for intermediate calculation
            d_intermediate_flat,// flattened memory for intermediate calculation
            Nsystems, // number of systems
            Neqn_p_sys); // number of equations in each system

        tnow+=timestep;

    }
    return nsteps;
}

int BDF2ErrorLoop(
    float tnow,
    float tend,
    float ** d_Jacobianss, // matrix (jacobian) input
    float * d_Jacobianss_flat,
    float * jacobian_zeros,
    float ** d_identity, // pointer to identity (ideally in constant memory?)
    float ** d_derivatives, // vector (derivatives) input
    float * d_derivatives_flat, // dy vector output
    float * equations,
    float * d_current_state_flat, // y vector output
    float * d_half_current_state_flat,
    float * d_previous_state_flat,
    float ** d_intermediate,
    float * d_intermediate_flat,
    float * d_constants,
    int Nsystems, // number of systems
    int Neqn_p_sys){

    float timestep = tend-tnow;
    int n_integration_steps = 2;

    int * error_flag = (int *) malloc(sizeof(int));
    int * d_error_flag;
    hipMalloc(&d_error_flag,sizeof(int));
    *error_flag = 0;
    hipMemcpy(d_error_flag,error_flag,sizeof(int),hipMemcpyHostToDevice);

/* -------------- configure the grid  ------------ */
    int threads_per_block = min(Neqn_p_sys,MAX_THREADS_PER_BLOCK);
    int x_blocks_per_grid = 1+Neqn_p_sys/MAX_THREADS_PER_BLOCK;
    int y_blocks_per_grid = min(Nsystems,MAX_BLOCKS_PER_GRID);
    int z_blocks_per_grid = 1+Nsystems/MAX_BLOCKS_PER_GRID;

    dim3 vector_gridDim(
        x_blocks_per_grid,
        y_blocks_per_grid,
        z_blocks_per_grid);

    dim3 ode_gridDim(
        1,
        y_blocks_per_grid,
        z_blocks_per_grid);
/* ----------------------------------------------- */
    
    // use a flag as a counter, why not
    int unsolved = 1;
    int nsteps=0;
    while (unsolved){
        nsteps+= BDF2SolveSystem(
            tnow,
            tend,
            timestep/n_integration_steps,
            d_Jacobianss,
            d_Jacobianss_flat,
            jacobian_zeros,
            d_identity,
            d_derivatives,
            d_derivatives_flat,
            d_current_state_flat,
            d_previous_state_flat,
            d_intermediate, // matrix memory for intermediate calculation
            d_intermediate_flat,// flattened memory for intermediate calculation
            d_constants,
            Nsystems,
            Neqn_p_sys);

        n_integration_steps*=2;

        nsteps+= BDF2SolveSystem(
            tnow,
            tend,
            timestep/n_integration_steps,
            d_Jacobianss,
            d_Jacobianss_flat,
            jacobian_zeros,
            d_identity,
            d_derivatives,
            d_derivatives_flat,
            d_half_current_state_flat,// the output state vector
            d_previous_state_flat,
            d_intermediate, // matrix memory for intermediate calculation
            d_intermediate_flat,// flattened memory for intermediate calculation
            d_constants,
            Nsystems,
            Neqn_p_sys);

        // determine if ANY of the INDEPENDENT systems are above the 
        //  the tolerance and fail them all. NOTE: this makes them not
        //  independent... 
        checkError<<<vector_gridDim,threads_per_block>>>(
            d_current_state_flat,d_half_current_state_flat,d_error_flag,
            Nsystems,Neqn_p_sys);

        // copy back the bool flag and determine if we done did it
        hipMemcpy(error_flag,d_error_flag,sizeof(int),hipMemcpyDeviceToHost);
        //*error_flag = 0;
        
        if (*error_flag){
            // increase the refinement level
            unsolved++;
            // put an upper limit on the refinement
            if (unsolved > 7){
                break;
            }

            printf("refining...%d\n",unsolved);
            *error_flag = 0;

            // reset the error flag on the device
            hipMemcpy(d_error_flag,error_flag,sizeof(int),hipMemcpyHostToDevice);
        
            // reset the equations
            hipMemcpy(
                d_current_state_flat,
                equations,
                Nsystems*Neqn_p_sys*sizeof(float),
                hipMemcpyHostToDevice);
            hipMemcpy(
                d_half_current_state_flat,
                equations,
                Nsystems*Neqn_p_sys*sizeof(float),
                hipMemcpyHostToDevice);
        }// if unsolved
        else{
            // we did it, let's exit the loop gracefully
            unsolved=0;
        }
    }// while unsolved
    return nsteps;
}

int cudaIntegrateBDF2(
    float tnow, // the current time
    float tend, // the time we integrating the system to
    float * constants, // the constants for each system
    float * equations, // a flattened array containing the y value for each equation in each system
    int Nsystems, // the number of systems
    int Neqn_p_sys){ // the number of equations in each system

    printf("BDF2 Received %d systems, %d equations per system\n",Nsystems,Neqn_p_sys);
    float *dest = equations;

    // define the identity matrix on the host
    float *identity_flat = (float *)malloc(Neqn_p_sys*Neqn_p_sys*sizeof(float));
    setIdentityMatrix(identity_flat,Neqn_p_sys);
    
    // set a batchsize of one
    float * d_identity_flat;
    float ** d_identity = initializeDeviceMatrix(identity_flat,&d_identity_flat,Neqn_p_sys*Neqn_p_sys,1);
    
/* -------------- move data to device ------------ */
    // zeros to initialize jacobians with
    float * jacobian_zeros = (float *) malloc(Nsystems*Neqn_p_sys*Neqn_p_sys*sizeof(float));
    for (int i=0; i<Neqn_p_sys*Neqn_p_sys*Nsystems; i++){
        jacobian_zeros[i]=0;
    }   

    // allocate memory for Jacobian matrices as a single "batch"
    float *d_Jacobianss_flat;
    float **d_Jacobianss = initializeDeviceMatrix(jacobian_zeros,&d_Jacobianss_flat,Neqn_p_sys*Neqn_p_sys,Nsystems);

    // initialize state-equation vectors
    float * zeros = (float *) malloc(Nsystems*Neqn_p_sys*sizeof(float));
    for (int i=0; i<Neqn_p_sys*Nsystems; i++){
        zeros[i]=0;
    }   

    // constants that define the ODEs
    /* TODO put this in constant memory instead-- does the below work? 
    __constant__ float d_constants[NUM_CONST]; // NUM_CONST #define'd in ode.h
    hipMemcpyToSymbol(HIP_SYMBOL(constants),d_constants,sizeof(d_constants));
    */
    float * d_constants;
    hipMalloc(&d_constants,NUM_CONST*sizeof(float));
    hipMemcpy(d_constants,constants,NUM_CONST*sizeof(float),hipMemcpyHostToDevice);

    // state equations, where output will be stored
    float *d_current_state_flat;
    float **d_current_state = initializeDeviceMatrix(equations,&d_current_state_flat,Neqn_p_sys,Nsystems);

    float *d_half_current_state_flat;
    float **d_half_current_state = initializeDeviceMatrix(
        equations,&d_half_current_state_flat,Neqn_p_sys,Nsystems);

    // saving previous step Y(n-1) because we need that for BDF2
    float *d_previous_state_flat;
    float **d_previous_state = initializeDeviceMatrix(zeros,&d_previous_state_flat,Neqn_p_sys,Nsystems);

    // memory for intermediate calculation... reuse it so we aren't constantly allocating
    //  and deallocating memory, NOTE can we remove this??
    float *d_intermediate_flat;
    float **d_intermediate = initializeDeviceMatrix(zeros,&d_intermediate_flat,Neqn_p_sys,Nsystems);

    // initialize derivative vectors
    float *d_derivatives_flat;
    float **d_derivatives = initializeDeviceMatrix(zeros,&d_derivatives_flat,Neqn_p_sys,Nsystems);

/* ----------------------------------------------- */

    int nsteps = BDF2ErrorLoop(
        tnow,
        tend,
        d_Jacobianss, // matrix (jacobian) input
        d_Jacobianss_flat,
        jacobian_zeros,
        d_identity, // pointer to identity (ideally in constant memory?)
        d_derivatives, // vector (derivatives) input
        d_derivatives_flat, // dy vector output
        equations,
        d_current_state_flat, // y vector output
        d_half_current_state_flat,
        d_previous_state_flat,
        d_intermediate, // matrix memory for intermediate calculation
        d_intermediate_flat,// flattened memory for intermediate calculation
        d_constants,
        Nsystems, // number of systems
        Neqn_p_sys);
    
    printf("nsteps taken: %d - tnow: %.2f\n",nsteps,tend);



/* -------------- copy data to host -------------- */
    // retrieve the output
    hipMemcpy(dest, d_half_current_state_flat, Neqn_p_sys*Nsystems*sizeof(float), hipMemcpyDeviceToHost);
/* ----------------------------------------------- */

/* -------------- shutdown by freeing memory   --- */
    hipFree(d_Jacobianss); hipFree(d_Jacobianss_flat);
    hipFree(d_current_state); hipFree(d_current_state_flat);
    hipFree(d_previous_state); hipFree(d_previous_state_flat);
    hipFree(d_intermediate); hipFree(d_intermediate_flat);
    hipFree(d_identity); hipFree(d_identity_flat);
    hipFree(d_derivatives); hipFree(d_derivatives_flat);

    free(zeros); free(jacobian_zeros);
    //free(temp_timestep);
    free(identity_flat);
/* ----------------------------------------------- */
    //return how many steps were taken
    return nsteps;
}
