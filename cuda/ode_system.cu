#include "hip/hip_runtime.h"

#include "ode.h"
#include "vector_kernels.h"
#include <stdio.h>

/* ---------------- CUDA Thread Block Organization ------------ */
void configureGrid(
    int Nsystems,int Neqn_p_sys,
    int * p_threads_per_block,
    dim3 * p_matrix_gridDim,
    dim3 * p_ode_gridDim,
    dim3 * p_vector_gridDim){

    int threads_per_block = min(Neqn_p_sys,MAX_THREADS_PER_BLOCK);
    int x_blocks_per_grid = 1+Neqn_p_sys/MAX_THREADS_PER_BLOCK;
    int y_blocks_per_grid = min(Nsystems,MAX_BLOCKS_PER_GRID);
    int z_blocks_per_grid = 1+Nsystems/MAX_BLOCKS_PER_GRID;

    dim3 matrix_gridDim(
        x_blocks_per_grid*Neqn_p_sys,
        y_blocks_per_grid,
        z_blocks_per_grid);

    dim3 ode_gridDim(
        1,
        y_blocks_per_grid,
        z_blocks_per_grid);

    dim3 vector_gridDim(
            x_blocks_per_grid,
            y_blocks_per_grid,
            z_blocks_per_grid);

    if (p_threads_per_block != NULL){
        *p_threads_per_block = threads_per_block;
    }

    if (p_matrix_gridDim != NULL){
        *p_matrix_gridDim = matrix_gridDim;
    }

    if (p_ode_gridDim != NULL){
        *p_ode_gridDim = ode_gridDim;
    }

    if (p_vector_gridDim != NULL){
        *p_vector_gridDim = vector_gridDim;
    }
}


/* ------------------------------------------------------------ */

__device__ int get_system_index(){
    return blockIdx.z*gridDim.y + blockIdx.y; 
}
__global__ void calculateDerivatives(
    float * d_derivatives_flat, 
    float * constants, 
    float * equations,
    int Nsystems,
    int Neqn_p_sys,
    float time){
    // isolate this system 

    int bid = get_system_index();
    // don't need to do anything, no system corresponds to this thread-block
    if (bid >= Nsystems){
        return;
    }

    int eqn_offset = bid*Neqn_p_sys;
    float * this_block_state = equations+eqn_offset;
    float * this_block_derivatives = d_derivatives_flat+eqn_offset;

    // eq. 16.6.1 in NR 
    this_block_derivatives[0] = 998.0*this_block_state[0] + 1998. * this_block_state[1];
    this_block_derivatives[1] = -999.0*this_block_state[0] - 1999.0*this_block_state[1];
}
__global__ void calculateJacobians(
    float **d_Jacobianss, 
    float * constants,
    float * equations,
    int Nsystems,
    int Neqn_p_sys,
    float time){

    // isolate this system 
    int bid = get_system_index();

    // don't need to do anything, no system corresponds to this thread-block
    if (bid >= Nsystems){
        return;
    }

    int eqn_offset = bid*Neqn_p_sys;
    float * this_block_state = equations+eqn_offset;
    float * this_block_jacobian = d_Jacobianss[bid];

    this_block_jacobian[0] = 998.0;
    this_block_jacobian[1] = -999.0;
    this_block_jacobian[2] = 1998.0;
    this_block_jacobian[3] = -1999.0;
}

void resetSystem(
    float ** d_derivatives,
    float * d_derivatives_flat,
    float ** d_Jacobianss,
    float * d_Jacobianss_flat,
    float * d_constants,
    float * d_current_state_flat,
    float * jacobian_zeros,
    int Nsystems,
    int Neqn_p_sys,
    float tnow){

    dim3 ode_gridDim;
    configureGrid(
        Nsystems,Neqn_p_sys,
        NULL,
        NULL,
        &ode_gridDim,
        NULL);


    if (d_derivatives_flat !=NULL){
        // evaluate the derivative function at tnow
        calculateDerivatives<<<ode_gridDim,1>>>(
            d_derivatives_flat,
            d_constants,
            d_current_state_flat,
            Nsystems,
            Neqn_p_sys,
            tnow);
    }

    if (d_Jacobianss_flat != NULL){
        // reset the jacobian, which has been replaced by (I-hJ)^-1
        hipMemcpy(
            d_Jacobianss_flat,jacobian_zeros,
            Nsystems*Neqn_p_sys*Neqn_p_sys*sizeof(float),
            hipMemcpyHostToDevice);

        calculateJacobians<<<ode_gridDim,1>>>(
            d_Jacobianss,
            d_constants,
            d_current_state_flat,
            Nsystems,
            Neqn_p_sys,
            tnow);
    }
}
