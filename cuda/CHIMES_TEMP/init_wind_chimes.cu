#include "hip/hip_runtime.h"
#include "chimes_vars.h"
#include "chimes_proto.h"

// link to global texture objects defined in wind_chimes.h
#include "wind_chimes.h"


void load_rate_coeffs_into_texture_memory(){
/* ------- chimes_table_constant ------- */
    // read the values from the corresponding chimes_table
    int N_reactions_all = chimes_table_constant.N_reactions[1];
    float * ratess_flat = chimes_table_constant.rates; // 1xN_reactions_all, not log

    // allocate the memory for the constant rates on the device
    hipMalloc(
        &wind_chimes_table_constant,
        sizeof(ChimesFloat)*N_reactions_all);

    // copy it over
    hipMemcpy(
        wind_chimes_table_constant,
        ratess_flat,
        sizeof(ChimesFloat)*N_reactions_all,
        hipMemcpyHostToDevice)

    // TODO need to copy over the reaction info
    N_reactions[0] and N_reactions[1] 
    reactantss // need to take the transpose
    productss // need to take the transpose

    H2_form_heating_reaction_index
        
/* ------- chimes_table_T_dependent ------- */
    // read the values from the corresponding chimes_table
    N_reactions_all = chimes_table_T_dependent.N_reactions[1];
    // put the flat pointer at the head of the 2d array
    ratess_flat = chimes_table_T_dependent.rates[0];

    // TODO need to loop over reactions and make sure
    //  rates are stored how I think they are in memory
    //  before copying them to Array TODO


    // allocate memory on device for these rates constants and 
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipMemcpy3DParms my_params = {0};
    //my_params.srcPos = make_hipPos(0,0,0);
    //my_params.dstPos = make_hipPos(0,0,0);
    my_params.srcPtr = make_hipPitchedPtr(ratess_flat,N_Temperature *sizeof(float),N_Temperature,1);
    my_params.kind = hipMemcpyHostToDevice;
    my_params.extent = make_hipExtent(N_Temperature, 1, N_reactions_all);

    // create the cuda array and copy the data to it
    hipArray *cu_3darray;
    hipMalloc3DArray(
        &cu_3darray,
        &channelDesc,
        make_hipExtent(N_Temperature, 0,N_reactions_all),
        hipArrayLayered);
    my_params.dstArray = cu_3darray;
    hipMemcpy3D(&my_params);

    // Describe the input array
    hipResourceDesc            resDesc;
    memset(&resDesc,0,sizeof(hipResourceDesc));

    resDesc.resType            = hipResourceTypeArray;
    resDesc.res.array.array    = cu_3darray;

    // Describe the output texture
    hipTextureDesc             texDesc;
    memset(&texDesc,0,sizeof(hipTextureDesc));
    texDesc.normalizedCoords = true;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.readMode = hipReadModeElementType;

    hipCreateTextureObject(
        &wind_chimes_table_T_dependent,
        &resDesc,
        &texDesc,
        NULL);

    // TODO need to copy over the reaction info
    N_reactions[0] and N_reactions[1]
    reactantss
    productss

    H2_collis_dissoci_heating_reaction_index
    H2_form_heating_reaction_index


    // for looping through mallocs on the device...
    //https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#dynamic-global-memory-allocation-and-operations
}

void init_chimes_wind(struct globalVariables myGlobalVars){

    // call the existing C routine...
    init_chimes_wind(myGlobalVars);

    load_rate_coeffs_into_texture_memory();
    

    
}

void init_chimes_wind_hardcoded(struct globalVariables myGlobalVars){
    // use hardcoded rates arrays: 

    load_rate_coeffs_into_texture_memory();
}
