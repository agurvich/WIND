// link to global texture objects defined in wind_chimes.h
#include "wind_chimes.h"
#include "config.h"
#include "ode.h"

// host structs
struct wind_chimes_table_bins_struct  h_wind_chimes_table_bins;
struct wind_chimes_constant_struct h_wind_chimes_table_constant;
struct wind_chimes_T_dependent_struct h_wind_chimes_table_T_dependent;
struct wind_chimes_recombination_AB_struct h_wind_chimes_table_recombination_AB;
struct RHS_input_struct h_wind_chimes_RHS_input;

// pointers for the device structs
struct wind_chimes_table_bins_struct * d_p_wind_chimes_table_bins;
struct wind_chimes_constant_struct * d_p_wind_chimes_table_constant;
struct wind_chimes_T_dependent_struct * d_p_wind_chimes_table_T_dependent;
struct wind_chimes_recombination_AB_struct * d_p_wind_chimes_table_recombination_AB;

// pointer for struct that contains pointers to the other structs
struct RHS_input_struct * d_p_wind_chimes_RHS_input; // defined in wind_chimes.h?

void checkCudaError(){
    hipError_t error = hipGetLastError();
    if (error != hipSuccess){
        printf("Error: %s \n",hipGetErrorString(error));
    }
}

void initialize_table_constant(
    struct wind_chimes_constant_struct * p_this_table,
    int * N_reactions,
    int * reactantss_transpose_flat,
    int * productss_transpose_flat,
    int H2_form_heating_reaction_index){

    // bind the simple stuff that's already allocated in the struct
    p_this_table->N_reactions[0] = N_reactions[0];
    p_this_table->N_reactions[1] = N_reactions[1];
    p_this_table->H2_form_heating_reaction_index=H2_form_heating_reaction_index; 

    // allocate and copy the reactants over, then bind to the host this_table structure
    int * d_reactantss_transpose_flat;
    hipMalloc(&d_reactantss_transpose_flat,
        sizeof(int)*3*p_this_table->N_reactions[1]);
    hipMemcpy(d_reactantss_transpose_flat,
        reactantss_transpose_flat,
        sizeof(int)*3*p_this_table->N_reactions[1],
        hipMemcpyHostToDevice);
    p_this_table->reactantss_transpose_flat = d_reactantss_transpose_flat; // needs to be a device array

    // allocate and copy the products over, then bind to the host this_table structure
    int * d_productss_transpose_flat;
    hipMalloc(
        &d_productss_transpose_flat,
        sizeof(int)*3*p_this_table->N_reactions[1]);
    hipMemcpy(
        d_productss_transpose_flat,
        productss_transpose_flat,
        sizeof(int)*3*p_this_table->N_reactions[1],
        hipMemcpyHostToDevice);
    p_this_table->productss_transpose_flat = d_productss_transpose_flat; // needs to be a device array
    // allocate and copy the rates over, then bind to the host table structure 
}


void initialize_table_T_dependent(
    struct wind_chimes_T_dependent_struct * p_this_table,
    int * N_reactions,
    int * reactantss_transpose_flat,
    int * productss_transpose_flat,
    int H2_collis_dissoc_heating_reaction_index,
    int H2_form_heating_reaction_index){

    // bind the simple stuff that's already allocated in the struct
    p_this_table->N_reactions[0] = N_reactions[0];
    p_this_table->N_reactions[1] = N_reactions[1];
    p_this_table->H2_collis_dissoc_heating_reaction_index=H2_collis_dissoc_heating_reaction_index;
    p_this_table->H2_form_heating_reaction_index=H2_form_heating_reaction_index; 

    // allocate and copy the reactants over, then bind to the host this_table structure
    int * d_reactantss_transpose_flat;
    hipMalloc(&d_reactantss_transpose_flat,
        sizeof(int)*3*p_this_table->N_reactions[1]);
    hipMemcpy(d_reactantss_transpose_flat,
        reactantss_transpose_flat,
        sizeof(int)*3*p_this_table->N_reactions[1],
        hipMemcpyHostToDevice);
    p_this_table->reactantss_transpose_flat = d_reactantss_transpose_flat; // needs to be a device array

    // allocate and copy the products over, then bind to the host this_table structure
    int * d_productss_transpose_flat;
    hipMalloc(
        &d_productss_transpose_flat,
        sizeof(int)*3*p_this_table->N_reactions[1]);
    hipMemcpy(
        d_productss_transpose_flat,
        productss_transpose_flat,
        sizeof(int)*3*p_this_table->N_reactions[1],
        hipMemcpyHostToDevice);
    p_this_table->productss_transpose_flat = d_productss_transpose_flat; // needs to be a device array
    // allocate and copy the rates over, then bind to the host table structure 
}

void initialize_table_recombination_AB(
    struct wind_chimes_recombination_AB_struct * p_this_table,
    int * N_reactions,
    int * reactantss_transpose_flat,
    int * productss_transpose_flat){

    // bind the simple stuff that's already allocated in the struct
    p_this_table->N_reactions[0] = N_reactions[0];
    p_this_table->N_reactions[1] = N_reactions[1];

    // allocate and copy the reactants over, then bind to the host this_table structure
    int * d_reactantss_transpose_flat;
    hipMalloc(&d_reactantss_transpose_flat,
        sizeof(int)*3*p_this_table->N_reactions[1]);
    hipMemcpy(d_reactantss_transpose_flat,
        reactantss_transpose_flat,
        sizeof(int)*3*p_this_table->N_reactions[1],
        hipMemcpyHostToDevice);
    p_this_table->reactantss_transpose_flat = d_reactantss_transpose_flat; // needs to be a device array

    // allocate and copy the products over, then bind to the host this_table structure
    int * d_productss_transpose_flat;
    hipMalloc(
        &d_productss_transpose_flat,
        sizeof(int)*p_this_table->N_reactions[1]);
    hipMemcpy(
        d_productss_transpose_flat,
        productss_transpose_flat,
        sizeof(int)*p_this_table->N_reactions[1],
        hipMemcpyHostToDevice);
    p_this_table->productss_transpose_flat = d_productss_transpose_flat; // needs to be a device array
}

void initialize_table_bins(
    struct wind_chimes_table_bins_struct * p_this_table){

    // bind the simple stuff that's already allocated in the struct
    p_this_table->N_Temperatures = chimes_table_bins.N_Temperatures;
    ChimesFloat * d_Temperatures;
    hipMalloc(
        &d_Temperatures,
        sizeof(ChimesFloat)*(p_this_table->N_Temperatures));

    hipMemcpy(
        d_Temperatures,
        chimes_table_bins.Temperatures,
        sizeof(ChimesFloat)*(p_this_table->N_Temperatures),
        hipMemcpyHostToDevice);

    p_this_table->Temperatures = d_Temperatures; // needs to be a device array
}


void load_rate_coeffs_into_texture_memory(
    hipTextureObject_t * p_texture,
    float * texture_edgess_flat,
    int n_layers, // N_reactions_all
    int n_texture_edges // n_texture_edgess
    ){

    // allocate memory on device for these rates constants and 
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(
        32, 0, 0, 0, hipChannelFormatKindFloat);
    hipMemcpy3DParms my_params = {0};
    //my_params.srcPos = make_hipPos(0,0,0);
    //my_params.dstPos = make_hipPos(0,0,0);
    my_params.srcPtr = make_hipPitchedPtr(
        texture_edgess_flat,
        n_texture_edges *sizeof(float),// size in bytes
        n_texture_edges, // size in elements
        1); // height dimensionality? (e.g. 1 for 2d?)

    my_params.kind = hipMemcpyHostToDevice;
    my_params.extent = make_hipExtent(
        n_texture_edges, // x dim
        1, // y dim
        n_layers); // z dim (layers)

    // create the cuda array and copy the data to it
    hipArray *cu_3darray;
    hipMalloc3DArray(
        &cu_3darray,
        &channelDesc,
        make_hipExtent(
            n_texture_edges, // x dim
            0, // y dim -- according to stack overflow this needs to be 0  even though above it is 1
            n_layers), // z dim
        hipArrayLayered);
    my_params.dstArray = cu_3darray;
    hipMemcpy3D(&my_params);
    checkCudaError();

    // Describe the input array
    hipResourceDesc            resDesc;
    memset(&resDesc,0,sizeof(hipResourceDesc));

    resDesc.resType            = hipResourceTypeArray;
    resDesc.res.array.array    = cu_3darray;

    // Describe the output texture
    hipTextureDesc             texDesc;
    memset(&texDesc,0,sizeof(hipTextureDesc));
    texDesc.normalizedCoords = false;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.readMode = hipReadModeElementType;

    // create the texture object at the global memory structure
    hipCreateTextureObject(
        p_texture,// was passed the pointer to the texture by reference
        &resDesc,
        &texDesc,
        NULL);
    checkCudaError();
}

void flatten_rates(
    ChimesFloat ** rates,
    float ** p_ratess_flat,
    int N_reactions,
    int N_Temperatures){

    for (int rxn_i=0; rxn_i < N_reactions; rxn_i++){
        for (int temp_i=0; temp_i < N_Temperatures; temp_i++){
            (*p_ratess_flat)[rxn_i*N_Temperatures + temp_i] = (float) rates[rxn_i][temp_i];
        }
    }
}

void tranpose_flatten_chemical_equations(
    int ** chem_indices,
    int ** p_chem_indices_transpose_flat,
    int N_reactions,
    int N_chems){

    for (int i_chem=0; i_chem<N_chems; i_chem++){
        for (int i_rxn=0; i_rxn<N_reactions; i_rxn++){
            (*p_chem_indices_transpose_flat)[i_chem*N_reactions+i_rxn] = chem_indices[i_rxn][i_chem];
        }
    }
}

void create_wind_chimes_structs(){
    float * ratess_flat;
    int * reactantss_transpose_flat;
    int * productss_transpose_flat;
    int N_reactions_all;
/* ------- chimes_table_constant ------- */
    N_reactions_all = chimes_table_constant.N_reactions[1];

    // allocate the pointers for this table
    //ratess_flat = (float *) malloc(sizeof(float)*N_reactions_all);
    reactantss_transpose_flat = (int *) malloc(sizeof(int)*3*N_reactions_all);
    productss_transpose_flat = (int *) malloc(sizeof(int)*3*N_reactions_all);

    tranpose_flatten_chemical_equations(
        chimes_table_constant.reactants,
        &reactantss_transpose_flat, 
        N_reactions_all,
        2); // 2 reactants per reaction

    tranpose_flatten_chemical_equations(
        chimes_table_constant.products,
        &productss_transpose_flat, 
        N_reactions_all,
        3); // 3 products per reaction

    // read the values from the corresponding chimes_table
    initialize_table_constant(
        &h_wind_chimes_table_constant,
        chimes_table_constant.N_reactions,
        reactantss_transpose_flat,
        productss_transpose_flat,
        chimes_table_constant.H2_form_heating_reaction_index);

    // allocate the memory for the constant rates on the device
    //  which are just an array, no interpolation required
    ChimesFloat * d_rates;
    hipMalloc(
        &d_rates,
        sizeof(ChimesFloat)*N_reactions_all);
    hipMemcpy(
        d_rates,
        chimes_table_constant.rates,
        sizeof(ChimesFloat)*N_reactions_all,
        hipMemcpyHostToDevice);
    h_wind_chimes_table_constant.rates = d_rates; // needs to be a device array

    // and free up the ratess, productss, and reactantss buffers
    free(reactantss_transpose_flat);
    free(productss_transpose_flat);
    //free(ratess_flat);

    // copy the host struct to the device
    hipMalloc(
        &d_p_wind_chimes_table_constant,
        sizeof(wind_chimes_constant_struct));
    hipMemcpy(
        d_p_wind_chimes_table_constant,
        &h_wind_chimes_table_constant,
        sizeof(wind_chimes_constant_struct),
        hipMemcpyHostToDevice);

/* ------- chimes_table_T_dependent ------- */
    N_reactions_all = chimes_table_T_dependent.N_reactions[1];

    // (re-)allocate the pointers for this table
    ratess_flat = (float *) malloc(
        sizeof(float)*
        chimes_table_bins.N_Temperatures*
        N_reactions_all);

    // flatten the reaction rates
    flatten_rates(
        chimes_table_T_dependent.rates,
        &ratess_flat,
        N_reactions_all,
        chimes_table_bins.N_Temperatures);

    reactantss_transpose_flat = (int *) malloc(sizeof(int)*3*N_reactions_all);
    productss_transpose_flat = (int *) malloc(sizeof(int)*3*N_reactions_all);

    tranpose_flatten_chemical_equations(
        chimes_table_T_dependent.reactants,
        &reactantss_transpose_flat, 
        N_reactions_all,
        3); // 3 reactants per reaction

    tranpose_flatten_chemical_equations(
        chimes_table_T_dependent.products,
        &productss_transpose_flat, 
        N_reactions_all,
        3); // 3 products per reaction

    // copy the values from the table over...
    initialize_table_T_dependent(
        &h_wind_chimes_table_T_dependent,
        chimes_table_T_dependent.N_reactions,
        reactantss_transpose_flat,
        productss_transpose_flat,
        chimes_table_T_dependent.H2_collis_dissoc_heating_reaction_index,
        chimes_table_T_dependent.H2_form_heating_reaction_index);

    // TODO need to make sure rates is in the right format TODO
    // read the rate coeffs from the corresponding chimes_table
    //  and put them into texture memory
    load_rate_coeffs_into_texture_memory(
        &h_wind_chimes_table_T_dependent.rates,
        ratess_flat,
        N_reactions_all,
        chimes_table_bins.N_Temperatures);

    // free up the ratess, productss, and reactantss buffers
    free(reactantss_transpose_flat);
    free(productss_transpose_flat);
    free(ratess_flat);

    // copy the host struct to the device
    hipMalloc(
        &d_p_wind_chimes_table_T_dependent,
        sizeof(wind_chimes_T_dependent_struct));
    hipMemcpy(
        d_p_wind_chimes_table_T_dependent,
        &h_wind_chimes_table_T_dependent,
        sizeof(wind_chimes_T_dependent_struct),
        hipMemcpyHostToDevice);

/* ------- chimes_table_recombination_AB ------- */
    N_reactions_all = chimes_table_recombination_AB.N_reactions[1];

    // (re-)allocate the pointers for this table
    ratess_flat = (float *) malloc(
        sizeof(float)*
        chimes_table_bins.N_Temperatures*
        N_reactions_all*
        2); // 2x the rates, one for A, one for B


    // flatten the case A rates
    flatten_rates(
        chimes_table_recombination_AB.rates[0],
        &(ratess_flat),
        N_reactions_all,
        chimes_table_bins.N_Temperatures);

    float * offset = ratess_flat + chimes_table_bins.N_Temperatures*N_reactions_all;
    // case B recombination is after case A
    flatten_rates(
        chimes_table_recombination_AB.rates[1],
        &offset,
        N_reactions_all,
        chimes_table_bins.N_Temperatures);


    reactantss_transpose_flat = (int *) malloc(sizeof(int)*3*N_reactions_all);
    productss_transpose_flat = (int *) malloc(sizeof(int)*3*N_reactions_all);

    tranpose_flatten_chemical_equations(
        chimes_table_recombination_AB.reactants,
        &reactantss_transpose_flat, 
        N_reactions_all,
        2); // 3 reactants per reaction

    // only 1 product per reaction -> this is already in the format we need!
    productss_transpose_flat = chimes_table_recombination_AB.products;

    // copy the values from the table over...
    initialize_table_recombination_AB(
        &h_wind_chimes_table_recombination_AB,
        chimes_table_recombination_AB.N_reactions,
        reactantss_transpose_flat,
        productss_transpose_flat);

    // TODO need to make sure rates is in the right format TODO
    // read the rate coeffs from the corresponding chimes_table
    //  and put them into texture memory
    load_rate_coeffs_into_texture_memory(
        &h_wind_chimes_table_recombination_AB.rates,
        ratess_flat,
        2*N_reactions_all, // 2x N_reactions_all layers, first half for A second for B
        chimes_table_bins.N_Temperatures);

    // for looping through mallocs on the device...
    //https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#dynamic-global-memory-allocation-and-operations

    // copy the host struct to the device
    hipMalloc(
        &d_p_wind_chimes_table_recombination_AB,
        sizeof(wind_chimes_recombination_AB_struct));
    hipMemcpy(
        d_p_wind_chimes_table_recombination_AB,
        &h_wind_chimes_table_recombination_AB,
        sizeof(wind_chimes_recombination_AB_struct),
        hipMemcpyHostToDevice);

/* ------- chimes_table_bins ------- */
    initialize_table_bins(&h_wind_chimes_table_bins);

    // copy the host struct to the device
    hipMalloc(
        &d_p_wind_chimes_table_bins,
        sizeof(wind_chimes_table_bins_struct));
    hipMemcpy(
        d_p_wind_chimes_table_bins,
        &h_wind_chimes_table_bins,
        sizeof(wind_chimes_table_bins_struct),
        hipMemcpyHostToDevice);


/* ------- RHS input struct ------- */
    // bind the pointers we just created to a single
    //  struct to pass them all at once
    h_wind_chimes_RHS_input.table_constant = d_p_wind_chimes_table_constant;
    h_wind_chimes_RHS_input.table_T_dependent = d_p_wind_chimes_table_T_dependent;
    h_wind_chimes_RHS_input.table_recombination_AB = d_p_wind_chimes_table_recombination_AB;
    h_wind_chimes_RHS_input.table_bins = d_p_wind_chimes_table_bins;

    // copy the host struct to the device
    hipMalloc(
        &d_p_wind_chimes_RHS_input,
        sizeof(RHS_input_struct));

    hipMemcpy(
        d_p_wind_chimes_RHS_input,
        &h_wind_chimes_RHS_input,
        sizeof(RHS_input_struct),
        hipMemcpyHostToDevice);

    RHS_input = (void *) d_p_wind_chimes_RHS_input;
}

// to unmangle the name, since I can
extern "C" {
    void init_wind_chimes(struct globalVariables * myGlobalVars){
        // call the existing C routine...
        printf("Initializing CHIMES...");
        init_chimes(myGlobalVars);
        printf("...finished initializing CHIMES!\n");

        printf("Copying data to GPU...");
        create_wind_chimes_structs();
        printf("...finished!\n");
    }
}

void init_chimes_wind_hardcoded(struct globalVariables myGlobalVars){
    // use hardcoded rates arrays: 
    create_wind_chimes_structs();
}
