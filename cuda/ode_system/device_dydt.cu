#include "hip/hip_runtime.h"
#include "device.h"
#include "ode.h"

__device__ float calculate_dydt(
    float tnow,
    float * constants,
    float * equations){
/* ----- PREFIX FLAG FOR PYTHON FRONTEND ----- */
    // constraint equation, ne = nH+ + nHe+ + 2*nHe++
    float ne = equations[1]+equations[3]+equations[4]*2.0;

    /* constants = [
        Gamma_(e,H0), Gamma_(gamma,H0), 
        alpha_(H+),
        Gamma_(e,He0), Gamma_(gamma,He0), 
        Gamma_(e,He+), Gamma_(gamma,He+),
        alpha_(He+),
        alpha_(d),
        alpha_(He++)
        ] 
    */ 


    if (threadIdx.x == 0){
        // H0 : alpha_(H+) ne nH+ - (Gamma_(e,H0)ne + Gamma_(gamma,H0))*nH0
        return constants[2]*ne*equations[1]
            -(constants[0]*ne + constants[1])*equations[0]; 
    }
    else if (threadIdx.x == 1){
        // H+ : (Gamma_(e,H0)ne + Gamma_(gamma,H0))*nH0 - alpha_(H+) ne nH+
        return -constants[2]*ne*equations[1]
            +(constants[0]*ne + constants[1])*equations[0]; 
    }
    else if (threadIdx.x == 2){
        // He0 :(alpha_(He+)+alpha_(d)) ne nHe+ - (Gamma_(e,He0)ne + Gamma_(gamma,He0)) nHe0
        return (constants[7]+constants[8])*ne*equations[3] 
            - (constants[3]*ne+constants[4])*equations[2];
    }
    else if (threadIdx.x == 3){
        // He+ : 
        //  alpha_(He++) ne nHe++ 
        //  + (Gamma_(e,He0)ne + Gamma_(gamma,He0)) nHe0
        //  - (alpha_(He+)+alpha_(d)) ne nHe+ 
        //  - (Gamma_(e,He+)ne + Gamma_(gamma,He+)) nHe+
        return constants[9]*ne*equations[4] 
            + (constants[3]*ne+constants[4])*equations[2]  
            - (constants[7]+constants[8])*ne*equations[3] 
            - (constants[5]*ne+constants[6])*equations[3];
    }
    else if (threadIdx.x == 4){
        // He++ : (5-Gamma_(e,He+)ne + 6-Gamma_(gamma,He+)) nHe+ 
        //  - 9-alpha_(He++) ne nHe++
        return (constants[5]*ne+constants[6])*equations[3]
        -constants[9]*ne*equations[4];
    }
/* ----- SUFFIX FLAG FOR PYTHON FRONTEND ----- */
    
   else{
        return NULL;
    } 
} // calculate_dydt

__device__ void calculate_jacobian(
    float tnow,
    float * constants,
    float * shared_temp_equations,
    float * Jacobian,
    int Neqn_p_sys){

    // constraint equation, ne = nH+ + nHe+ + 2*nHe++
    float ne = shared_temp_equations[1]+shared_temp_equations[3]+shared_temp_equations[4]*2.0;

    /* constants = [
        0-Gamma_(e,H0), 1-Gamma_(gamma,H0), 
        2-alpha_(H+),
        3-Gamma_(e,He0), 4-Gamma_(gamma,He0), 
        5-Gamma_(e,He+), 6-Gamma_(gamma,He+),
        7-alpha_(He+),
        8-alpha_(d),
        9-alpha_(He++)
        ] 
    */

   
    // NOTE could make this faster if we could do it in paralell 
    if (threadIdx.x == 0){
/* ----- PREFIX FLAG FOR PYTHON FRONTEND ----- */
        // H0
        Jacobian[0] = -(constants[0]*ne + constants[1]); // H+ : -(Gamma_(e,H0)ne + Gamma_(gamma,H0))
        Jacobian[1] = -Jacobian[0]; // H0 : 0-Gamma_(e,H0)ne + 1-Gamma_(gamma,H0)
            
        //H+
        Jacobian[6] = -constants[2]*ne; // H+ -alpha_(H+)ne
        Jacobian[5] = -Jacobian[6]; // H0 : 2-alpha_(H+)ne
            
        // He0
        Jacobian[12] = -(constants[3]*ne+constants[4]); //He0 : -(Gamma_(e,He0)ne + Gamma_(gamma,He0))
        Jacobian[13] = Jacobian[12]; //He+ : 3-Gamma_(e,He0)ne + 4-Gamma_(gamma,He0)
            
        // He+
        Jacobian[19] = constants[5]*ne+constants[6]; //He++ : 5-Gamma_(e,He+)ne + 6-Gamma_(gamma,He+)
        Jacobian[17] = (constants[7]+constants[8])*ne; //He0 : (7-alpha_(He+)+8-alpha_(d))ne
        Jacobian[18] = -Jacobian[17] - 
            Jacobian[19]; //He+ : -((alpha_(He+)+alpha_(d)+Gamma_(e,He+))ne+Gamma_(gamma,He+))
            
        // He++
        Jacobian[24] = -constants[9]*ne;//He++ : -alpha_(He++)ne
        Jacobian[23] = -Jacobian[24];//He+ : 9-alpha_(He++)ne
/* ----- SUFFIX FLAG FOR PYTHON FRONTEND ----- */
    }

    __syncthreads();
} //calculate_jacobian 
