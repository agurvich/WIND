
#include <hip/hip_runtime.h>
#include <stdio.h>
// Simple transformation kernel
__global__ void transformKernel(
    float* d_output,
    hipTextureObject_t texObj,
    int width){
    // Calculate normalized texture coordinates
    float u = threadIdx.x/(float) blockDim.x;
    // Read from texture and write to global memory
    d_output[threadIdx.x] = tex1D<float>(texObj,u);

    for (int i = 0; i < blockDim.x; i++){
        if (threadIdx.x == i){
            printf("(%.2f, %.2f )\t",u,d_output[threadIdx.x]);
        }
        __syncthreads();
    }
    __syncthreads();
    if (threadIdx.x == 1){
        printf("\n");
    }
}

// Host code
int main(){
    int width = 10;
    float * h_data =(float *) malloc(sizeof(float)*width);;
    for (int i=0; i<width; i++){
        h_data[i]=(float) i;
    }
    int size = width*sizeof(float);

    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(
        32, 0, 0, 0,
        hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, size,1);
    printf("cuda malloc array\n");
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    // Copy to device memory some data located at address h_data
    // in host memory 
    hipMemcpyToArray(
        cuArray, 0, 0,
        h_data, size,
        hipMemcpyHostToDevice);
    printf("cuda memcpy to array\n");
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    // Create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(
        &texObj,
        &resDesc,
        &texDesc,
        NULL);
    printf("create texture \n");
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    int num = 10;
    // Allocate result of transformation in device memory
    float* d_output;
    hipMalloc(&d_output, num*width*sizeof(float));
    float* output = (float *) malloc(num*width*sizeof(float));

    // Invoke kernel
    transformKernel<<<1,100>>>(
        d_output,
        texObj, width);
    
    // retrieve the output
    hipMemcpy(output,d_output,num*width*sizeof(float),hipMemcpyDeviceToHost);

    for (int i=0; i< width; i++){
        printf("%.2f \t",h_data[i]);
    }
    printf("\n");

    // Destroy texture object
    hipDestroyTextureObject(texObj);

    // Free device memory
    hipFreeArray(cuArray);
    hipFree(d_output);

    return 0;
}
