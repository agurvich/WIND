#include "hip/hip_runtime.h"
#include <stdio.h>

texture<float, 1, hipReadModeElementType> tex;

__global__ void kernel(
    int texture_size,
    float* normalized_indices){
    for (int i = 0; i < blockDim.x; i++){
        if (threadIdx.x == i){
            float v=0.5+normalized_indices[threadIdx.x]*(
                texture_size-1);
            float x = tex1D(tex, v);
            printf("(%.2f, %.2f, %.2f)\n",
                normalized_indices[threadIdx.x],
                v-0.5,x);
        }
        __syncthreads();
    }
    __syncthreads();
    if (threadIdx.x == 1){
        printf("\n");
    }
}

int main(){
    int ntexture_edges = 6; // how many "anchors" are in the texture
    int nsamples = 10; // sample the texture in 1/nsamples increments

    // create the normalized_indices
    //  +1 to get the final 1.0 at the end
    float *normalized_indices = (float*)malloc(
        (nsamples+1)*sizeof(float));

    for (int i=0; i<(nsamples+1); i++){
        normalized_indices[i] = float(i)/float(nsamples);
    }
    normalized_indices[3]=.75; // overwrite to check manually

    // create the device normalized_indices pointer and fill it
    float * d_normalized_indices;
    hipMalloc((void**)&d_normalized_indices,
        sizeof(float)*(nsamples+1));
    hipMemcpy(d_normalized_indices,
        normalized_indices,
        sizeof(float)*(nsamples+1),
        hipMemcpyHostToDevice);

    // fill array with texture values
    float *data = (float*)malloc(ntexture_edges*sizeof(float));
    for (int i = 0; i < ntexture_edges; i++){
        data[i] = 2*float(i);
        printf("%d\t",i);
    }
    printf("\n");

    // make an array to store the texture values
    hipArray* cuArray;

    hipMallocArray(&cuArray, &tex.channelDesc, ntexture_edges, 1);
    hipBindTextureToArray (tex, cuArray);
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = 0;


    hipMemcpyToArray(cuArray,0,0,data,sizeof(float)*ntexture_edges,
        hipMemcpyHostToDevice);

    kernel<<<1, nsamples+1>>>(
        ntexture_edges,
        d_normalized_indices);

    hipDeviceSynchronize();

    free(data);
    free(normalized_indices);

    hipFreeArray(cuArray);
    hipFree(d_normalized_indices);
}



