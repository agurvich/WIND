#include "hip/hip_runtime.h"
#include <stdio.h>

texture<float, 1, hipReadModeElementType> tex;

__global__ void kernel(int M, int N, float *d_out){
    float v = 0.5+ float(threadIdx.x) /float(N+1)* float(M);
    float x = tex1D(tex, v);
    //printf("%f\n", x); // for deviceemu testing
    d_out[threadIdx.x] = x;

    for (int i = 0; i < blockDim.x; i++){
        if (threadIdx.x == i){
            printf("(%.2f, %.2f )\t",v-0.5,d_out[threadIdx.x]);
        }
        __syncthreads();
    }
    __syncthreads();
    if (threadIdx.x == 1){
        printf("\n");
    }


    }

int main(){
    int M = 2;
    int nbins = 20;
    int N = M*nbins-1;

    // memory for output

    float *d_out;

    hipMalloc((void**)&d_out, sizeof(float) * N);



    // make an array half the size of the output

    hipArray* cuArray;

    hipMallocArray(&cuArray, &tex.channelDesc, M, 1);
    hipBindTextureToArray (tex, cuArray);

    tex.filterMode = hipFilterModeLinear;
    tex.normalized = 0;

    // data fill array with increasing values
    float *data = (float*)malloc(M*sizeof(float));

    for (int i = 0; i < M; i++)
        data[i] = float(i);
    ( hipMemcpyToArray(cuArray, 0, 0, data, sizeof(float)*M, hipMemcpyHostToDevice) );



    kernel<<<1, nbins+1>>>(M,N, d_out);

    float *h_out = (float*)malloc(sizeof(float)*N);
    ( hipMemcpy(h_out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost) );
    /*
    for (int i = 0; i < N; i++)
        printf("%f\n", h_out[i]);
    */

    free(h_out);
    free(data);

    hipFreeArray(cuArray);
    hipFree(d_out);
}



