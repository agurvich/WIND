
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernelSampleTexture(
    hipTextureObject_t tex,
    int texture_size,
    float* normalized_indices){

    // force threads to execute in a specific order
    for (int i = 0; i < blockDim.x; i++){
        if (threadIdx.x == i){
            float v=0.5+(normalized_indices[threadIdx.x]*
                (texture_size-1));
            float x = tex1D<float>(tex, v);
            printf("(%.2f, %.2f, %.2f)\n",
                normalized_indices[threadIdx.x],
                v-0.5,x);
        }
        __syncthreads();
    }
}

hipTextureObject_t make1DTextureFromPointer(
    float * arr,
    int Narr){
    
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(
        32,0,0,0,
        hipChannelFormatKindFloat);
    hipArray* cuArray;
    // cudaMallocArray inherently allocates 2D, 
    //  last argument is 
    hipMallocArray(
        &cuArray,
        &channelDesc,
        Narr,
        1);

    // cudaMemcpyToArray is deprecated for some reason...
    //  so we're supposed to be using Memcpy2DToArray
    //  https://devtalk.nvidia.com/default/topic/1048376/cuda-programming-and-performance/cudamemcpytoarray-is-deprecated/
    hipMemcpyToArray(
        cuArray, // destination of data
        0,0, // woffset and hoffset?
        arr, // source of data
        sizeof(float)*Narr, // bytes of data
        hipMemcpyHostToDevice);

    //create texture object
    struct hipResourceDesc resDesc;
    memset(&resDesc,0,sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] =hipAddressModeClamp;
    texDesc.addressMode[1] =hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // create texture object: we only have to do this once!
    hipTextureObject_t tex=0;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
    return tex;
}

void sampleTexture(
    hipTextureObject_t tex,
    int ntexture_edges,
    int nsamples){

    // create the normalized_indices
    //  +1 to get the final 1.0 at the end
    float *normalized_indices = (float*)malloc(
        (nsamples+1)*sizeof(float));

    for (int i=0; i<(nsamples+1); i++){
        normalized_indices[i] = float(i)/float(nsamples);
    }

    // create the device normalized_indices pointer and fill it
    float * d_normalized_indices;
    hipMalloc((void**)&d_normalized_indices,
        sizeof(float)*(nsamples+1));
    hipMemcpy(d_normalized_indices,
        normalized_indices,
        sizeof(float)*(nsamples+1),
        hipMemcpyHostToDevice);
    kernelSampleTexture<<<1, nsamples+1>>>(
        tex,
        ntexture_edges,
        d_normalized_indices);

    hipDeviceSynchronize();
    free(normalized_indices);
    hipFree(d_normalized_indices);
}

int main(){
    int ntexture_edges = 6; // how many "anchors" are in the texture
    int nsamples = 10; // sample the texture in 1/nsamples increments

    // fill array with texture values
    float *data = (float*)malloc(ntexture_edges*sizeof(float));
    for (int i = 0; i < ntexture_edges; i++){
        data[i] = 2*float(i);
        printf("%d\t",i);
    }
    printf("\n");
    
    hipTextureObject_t tex = make1DTextureFromPointer(
        data,ntexture_edges);

    sampleTexture(tex,ntexture_edges,nsamples);

    free(data);
    hipDestroyTextureObject(tex);
    //cudaFree(cuArray); <--- rip?
}
