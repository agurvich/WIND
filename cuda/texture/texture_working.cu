
#include <hip/hip_runtime.h>
#include <stdio.h>

//texture<float, 1, cudaReadModeElementType> tex;

__global__ void kernel(
    hipTextureObject_t tex,
    int texture_size,
    float* normalized_indices){
    for (int i = 0; i < blockDim.x; i++){
        if (threadIdx.x == i){
            float v=0.5+normalized_indices[threadIdx.x];
            float x = tex1D<float>(tex, v);
            printf("(%.2f, %.2f, %.2f)\n",
                normalized_indices[threadIdx.x],
                v-0.5,x);
        }
        __syncthreads();
    }
    __syncthreads();
    if (threadIdx.x == 1){
        printf("\n");
    }
}

int main(){
    int ntexture_edges = 6; // how many "anchors" are in the texture
    int nsamples = 10; // sample the texture in 1/nsamples increments

    // create the normalized_indices
    //  +1 to get the final 1.0 at the end
    float *normalized_indices = (float*)malloc(
        (nsamples+1)*sizeof(float));

    for (int i=0; i<(nsamples+1); i++){
        normalized_indices[i] = float(i)/float(nsamples);
    }
    normalized_indices[3]=.75; // overwrite to check manually

    // create the device normalized_indices pointer and fill it
    float * d_normalized_indices;
    hipMalloc((void**)&d_normalized_indices,
        sizeof(float)*(nsamples+1));
    hipMemcpy(d_normalized_indices,
        normalized_indices,
        sizeof(float)*(nsamples+1),
        hipMemcpyHostToDevice);

    // fill array with texture values
    float *data = (float*)malloc(ntexture_edges*sizeof(float));
    for (int i = 0; i < ntexture_edges; i++){
        data[i] = 2*float(i);
        printf("%d\t",i);
    }
    printf("\n");

    float *d_data;
    hipMalloc((void**)&d_data,
        sizeof(float)*(ntexture_edges));
    hipMemcpy(d_data,data,ntexture_edges*sizeof(float),
        hipMemcpyHostToDevice);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(
        32,0,0,0,
        hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, ntexture_edges, 1);

    hipMemcpyToArray(cuArray,0,0,data,sizeof(float)*ntexture_edges,
        hipMemcpyHostToDevice);

    //create texture object
    struct hipResourceDesc resDesc;
    memset(&resDesc,0,sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    //resDesc.res.linear.devPtr = d_data;
    //resDesc.res.linear.desc = cudaCreateChannelDesc<float>(); 
    //resDesc.res.linear.sizeInBytes = sizeof(float)*(ntexture_edges);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] =hipAddressModeClamp;
    texDesc.addressMode[1] =hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

      // create texture object: we only have to do this once!
    hipTextureObject_t tex=0;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
    

    /*
    cudaMemcpy(d_normalized_indices,
        normalized_indices,
        sizeof(float)*(ntexture_edges),
        cudaMemcpyHostToDevice);
        */

    // fill array with texture values
    /*
    // make an array to store the texture values

    cudaBindTextureToArray (tex, cuArray);
    tex.filterMode = cudaFilterModeLinear;
    tex.normalized = 0;

    */

    kernel<<<1, nsamples+1>>>(
        tex,
        ntexture_edges,
        d_normalized_indices);

    hipDeviceSynchronize();

    free(data);
    free(normalized_indices);

    hipDestroyTextureObject(tex);
    //cudaFreeArray(cuArray);
    hipFree(d_normalized_indices);
}



