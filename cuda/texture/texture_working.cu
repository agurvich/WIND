
#include <hip/hip_runtime.h>
#include <stdio.h>

//texture<float, 1, cudaReadModeElementType> tex;

__global__ void kernel(
    hipTextureObject_t tex,
    int texture_size,
    float* normalized_indices){
    for (int i = 0; i < blockDim.x; i++){
        if (threadIdx.x == i){
            float v=0.5+(normalized_indices[threadIdx.x]*
                (texture_size-1));
            float x = tex1D<float>(tex, v);
            printf("(%.2f, %.2f, %.2f)\n",
                normalized_indices[threadIdx.x],
                v-0.5,x);
        }
        __syncthreads();
    }
    __syncthreads();
    if (threadIdx.x == 1){
        printf("\n");
    }
}

hipTextureObject_t make1DTextureFromPointer(
    float * arr,
    int Narr){
    
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(
        32,0,0,0,
        hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, Narr, 1);

    hipMemcpyToArray(cuArray,0,0,arr,sizeof(float)*Narr,
        hipMemcpyHostToDevice);

    //create texture object
    struct hipResourceDesc resDesc;
    memset(&resDesc,0,sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] =hipAddressModeClamp;
    texDesc.addressMode[1] =hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

      // create texture object: we only have to do this once!
    hipTextureObject_t tex=0;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
    return tex;
}

int main(){
    int ntexture_edges = 6; // how many "anchors" are in the texture
    int nsamples = 10; // sample the texture in 1/nsamples increments

    // create the normalized_indices
    //  +1 to get the final 1.0 at the end
    float *normalized_indices = (float*)malloc(
        (nsamples+1)*sizeof(float));

    for (int i=0; i<(nsamples+1); i++){
        normalized_indices[i] = float(i)/float(nsamples);
    }
    normalized_indices[3]=.75; // overwrite to check manually

    // create the device normalized_indices pointer and fill it
    float * d_normalized_indices;
    hipMalloc((void**)&d_normalized_indices,
        sizeof(float)*(nsamples+1));
    hipMemcpy(d_normalized_indices,
        normalized_indices,
        sizeof(float)*(nsamples+1),
        hipMemcpyHostToDevice);

    // fill array with texture values
    float *data = (float*)malloc(ntexture_edges*sizeof(float));
    for (int i = 0; i < ntexture_edges; i++){
        data[i] = 2*float(i);
        printf("%d\t",i);
    }
    printf("\n");
    
    hipTextureObject_t tex = make1DTextureFromPointer(
        data,ntexture_edges);

    kernel<<<1, nsamples+1>>>(
        tex,
        ntexture_edges,
        d_normalized_indices);

    hipDeviceSynchronize();

    free(data);
    free(normalized_indices);

    hipDestroyTextureObject(tex);
    //cudaFreeArray(cuArray);
    hipFree(d_normalized_indices);
}



